#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "problem2.h"

#define BLOCK_DIM 32

__global__ void mat_mult_optimized(double *dev_a, double *dev_b, double *dev_c, int ROW_A, int COL_A, int ROW_B, int COL_B)
{

	__shared__ double s_a[BLOCK_DIM][BLOCK_DIM];
	__shared__ double s_b[BLOCK_DIM][BLOCK_DIM];

	unsigned int row = BLOCK_DIM * blockIdx.y + threadIdx.y;
	unsigned int col = BLOCK_DIM * blockIdx.x + threadIdx.x;
	unsigned int i, j;

	double c = 0.0;

	#pragma unroll
	for (i = 0; i < (BLOCK_DIM + COL_A - 1) / BLOCK_DIM; i++)
	{
		if ((i * BLOCK_DIM + threadIdx.x < COL_A) && (row < ROW_A))
		{
			s_a[threadIdx.y][threadIdx.x] = dev_a[(row * COL_A) + (i * BLOCK_DIM) + threadIdx.x];
		} 
		else 
		{
			s_a[threadIdx.y][threadIdx.x] = 0.0;
		}

		if ((i * BLOCK_DIM + threadIdx.y < ROW_B) && (col < COL_B))
		{
			s_b[threadIdx.y][threadIdx.x] = dev_b[col + COL_B * (i * BLOCK_DIM + threadIdx.y)];
		}
		else
		{
			s_b[threadIdx.y][threadIdx.x] = 0.0;
		}
		__syncthreads();

		for (j = 0; j < BLOCK_DIM; j++)
		{
			c += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
		}
		__syncthreads();
	}

	if ((row < ROW_A) && (col < COL_B))
	{
		dev_c[(blockIdx.y * blockDim.y + threadIdx.y) * COL_B + (blockIdx.x * blockDim.x) + threadIdx.x] = c;
	}
}


__global__ void mat_mult_basic(double *dev_a, double *dev_b, double *dev_c, int ROW_A, int COL_A, int ROW_B, int COL_B)
{

	// Naive Matrix Multiplication

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	double sum = 0.0;
	int i;

	if ((row < ROW_A) && (col  < COL_B))
	{
		for (i = 0; i < COL_A; i++) 
		{
			sum += dev_a[row * COL_A + i] * dev_b[i * ROW_B + col];
		}
	dev_c[row * COL_B + col] = sum;
	}
}


double * mat_mult_gpu_basic(double * A, double * B, int ROW_A, int COL_A,  int ROW_B, int COL_B)
{

	int size_a, size_b, size_c;
	
	size_a = ROW_A * COL_A * sizeof(double);
	size_b = ROW_B * COL_B * sizeof(double);
	size_c = ROW_A * COL_B * sizeof(double);

	double *dev_a, *dev_b, *dev_c;
	
	double *C = (double *) malloc(size_c);
	hipMalloc((void**)&dev_a, size_a);	
	hipMalloc((void**)&dev_b, size_b);	
	hipMalloc((void**)&dev_c, size_c);	

	hipMemcpy(dev_a, A, size_a, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, size_b, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 blocksPerGrid((int)ceil((double)ROW_A / (double)threadsPerBlock.x), (int)ceil((double)COL_B / (double)threadsPerBlock.y));

	printf("Basic  GPU Matrix Multiplication\n");

	clock_t start, end;
	start = clock();

	mat_mult_basic<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, ROW_A, COL_A, ROW_B, COL_B);
	hipDeviceSynchronize();

	end = clock();
	printf("Time for Basic Matrix Multiplication: %f\n", ((double)end - (double)start) / CLOCKS_PER_SEC);

	hipMemcpy(C, dev_c, size_c, hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return C;
}


double * mat_mult_gpu_cublas(double * A, double * B, int ROW_A, int COL_A, int ROW_B, int COL_B)
{

	int size_a, size_b, size_c;
	
	size_a = ROW_A * COL_A * sizeof(double);
	size_b = ROW_B * COL_B * sizeof(double);
	size_c = ROW_A * COL_B * sizeof(double);

	double *dev_a, *dev_b, *dev_c;

	double *C = (double *) malloc(size_c);
	hipMalloc((void**)&dev_a, size_a);	
	hipMalloc((void**)&dev_b, size_b);	
	hipMalloc((void**)&dev_c, size_c);	

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasSetMatrix(ROW_A, COL_A, sizeof(double), A, ROW_A, dev_a, ROW_A);
	hipblasSetMatrix(ROW_B, COL_B, sizeof(double), B, ROW_B, dev_b, ROW_B);
	hipblasSetMatrix(ROW_A, COL_B, sizeof(double), C, ROW_A, dev_c, ROW_A);

	printf("CUBLAS GPU Matrix Multiplication\n");
	double alpha = 1.0;
	double beta = 1.0;

	clock_t start, end;
	start = clock();

	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ROW_A, COL_B, COL_A, &alpha, dev_a, ROW_A, dev_b, ROW_B, &beta, dev_c, ROW_A);
	hipDeviceSynchronize();

	end = clock();
	printf("Time for CUBLAS Matrix Multiplication: %f\n", ((double)end - (double)start) / CLOCKS_PER_SEC);

	hipblasGetMatrix(ROW_A, COL_B, sizeof(double), dev_c, ROW_A, C, ROW_A);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipblasDestroy(handle);

	return C;

}



double * mat_mult_gpu_optimized(double * A, double * B, int ROW_A, int COL_A,  int ROW_B, int COL_B)
{

	int size_a, size_b, size_c;
	
	size_a = ROW_A * COL_A * sizeof(double);
	size_b = ROW_B * COL_B * sizeof(double);
	size_c = ROW_A * COL_B * sizeof(double);

	double *dev_a, *dev_b, *dev_c;
	
	double *C = (double *) malloc(size_c);
	hipMalloc((void**)&dev_a, size_a);	
	hipMalloc((void**)&dev_b, size_b);	
	hipMalloc((void**)&dev_c, size_c);	

	hipMemcpy(dev_a, A, size_a, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, size_b, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 blocksPerGrid((int)ceil((double)ROW_A / (double)threadsPerBlock.x), (int)ceil((double)COL_B / (double)threadsPerBlock.y));

	printf("Optimized GPU Matrix Multiplication\n");
	
	clock_t start, end;
	start = clock();

	mat_mult_optimized<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, ROW_A, COL_A, ROW_B, COL_B);
	hipDeviceSynchronize();

	end = clock();
	printf("Time for Optimized Matrix Multiplication: %f\n", ((double)end - (double)start) / CLOCKS_PER_SEC);
	
	hipMemcpy(C, dev_c, size_c, hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return C;
}
