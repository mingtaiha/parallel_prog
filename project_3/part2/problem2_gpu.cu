#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "problem2.h"

#define BLOCK_DIM 32

__global__ void mat_mult_basic(double *dev_a, double *dev_b, double *dev_c, int ROW_A, int COL_A, int ROW_B, int COL_B)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	double sum = 0.0;
	int i;

	if ((row < ROW_A) && (col  < COL_B))
	{
		for (i = 0; i < COL_A; i++) 
		{
			sum += dev_a[row * COL_A + i] * dev_b[i * ROW_B + col];
		}
	dev_c[row * COL_B + col] = sum;
	}
}


double * mat_mult_gpu(double * A, double * B, int ROW_A, int COL_A,  int ROW_B, int COL_B)
{

	int size_a, size_b, size_c;
	
	size_a = ROW_A * COL_A * sizeof(double);
	size_b = ROW_B * COL_B * sizeof(double);
	size_c = ROW_A * COL_B * sizeof(double);

	double *dev_a, *dev_b, *dev_c;
	
	double *C = (double *) malloc(size_c);
	hipMalloc((void**)&dev_a, size_a);	
	hipMalloc((void**)&dev_b, size_b);	
	hipMalloc((void**)&dev_c, size_c);	

	hipMemcpy(dev_a, A, size_a, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, size_b, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 blocksPerGrid((int)ceil((double)ROW_A / (double)threadsPerBlock.x), (int)ceil((double)COL_B / (double)threadsPerBlock.y));

	printf("Basic GPU Matrix Multiplication\n");
	mat_mult_basic<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, ROW_A, COL_A, ROW_B, COL_B);
	hipDeviceSynchronize();

	hipMemcpy(C, dev_c, size_c, hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return C;
}
