#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "sm_20_atomic_functions.h"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } else { \
			printf("success\n"); \
		} \
    } while (0)


#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
int atomicOr(int*, int);
#endif

typedef struct vertex {
	int val;
	int vflag;
	int coordX, coordY;
} vertex;

typedef struct Node {
	int val;
	int* coord; //used to find value on graph array
	struct Node* next;
} Node;

//pop from the queue
__device__ Node queuePop(Node** headPointer) {
	Node* holder = *headPointer; //holds value to be referenced later

	if (*headPointer == NULL) { //if the link list is empty
		printf("Nothing to pop!\n");
		Node returnVal;
		returnVal.val = -1;
		returnVal.coord = NULL;
		return returnVal;
	}

	if ((*headPointer)->next == NULL) { //if there was only 1 element in the link list
		Node returnVal;
		returnVal.val = (*headPointer)->val;
		returnVal.coord = (*headPointer)->coord;
		*headPointer = NULL;
		free(holder);
		return returnVal;
	}

	Node* iter = (*headPointer)->next;
	while (1) {
		if (iter->next == NULL) {
			(*headPointer)->next = NULL;
			Node returnVal;
			returnVal.val = iter->val;
			returnVal.coord = iter->coord;
			free(iter);
			*headPointer = holder;
			return returnVal;
		}
		*headPointer = iter; //holder always stays 1 step behind iter
		iter = iter->next;
	}
}

__device__ void queuePush(Node** headPointer, int pushVal, int* pushCoord) { //set pushCoord to NULL if you dont care about coord value
	Node* pushNode = (Node*)malloc(sizeof(Node));
	pushNode->val = pushVal;
	pushNode->coord = pushCoord;
	if (*headPointer == NULL) { //if there is nothing in the queue initially
		pushNode->next = NULL;
	}
	else {
		pushNode->next = *headPointer;
	}
	*headPointer = pushNode;
	return;
}


//graph.c functions

void printGraph(vertex** graph, int size) {

	printf("\nGraph Values\n\n");
	int i = 0;
	for (; i < size; ++i) {
		int j = 0;
		for (; j < size; ++j) {
			if (graph[i][j].val != -1) {
				if (graph[i][j].val < 10) {
					printf("%d  ", graph[i][j].val);
				}
				else {
					printf("%d ", graph[i][j].val);
				}
			}
			else {
				printf("   ");
			}
		}
		printf("\n");
	}

	printf("\n");

	printf("Visited Flag Values\n\n");
	for (i = 0; i < size; ++i) {
		int j = 0;
		for (; j < size; ++j) {
			if (graph[i][j].val != -1) {
				printf("%d ", graph[i][j].vflag);
			}
			else {
				printf("  ");
			}
		}
		printf("\n");
	}
}

vertex** checkGraph(vertex** g, int size) { //for simplicity, change every value around isolated value to (not -1)
	vertex** graph;
	graph = g;
	int i = 0;
	for (; i < size; ++i) {
		int j = 0;
		for (; j < size; ++j) {
			if (i == 0) { //top row
				if (j == 0) { //top left corner
					if (graph[i + 1][j].val == -1 && graph[i][j + 1].val == -1 && graph[i + 1][j + 1].val == -1) {
						graph[i + 1][j].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i + 1][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
					}
				}
				else if (j == size - 1) { //top right corner
					if (graph[i + 1][j].val == -1 && graph[i][j - 1].val == -1 && graph[i + 1][j - 1].val == -1) {
						graph[i + 1][j].val = rand() % 100;
						graph[i][j - 1].val = rand() % 100;
						graph[i + 1][j - 1].val = rand() % 100;
					}
				}
				else { //just on top row
					if (graph[i][j - 1].val == -1 && graph[i][j + 1].val == -1 &&
						graph[i + 1][j - 1].val == -1 && graph[i + 1][j].val == -1 && graph[i + 1][j + 1].val == -1) {

						graph[i][j - 1].val = rand() % 100;
						graph[i][j + 1].val = rand() % 100;
						graph[i + 1][j - 1].val = rand() % 100;
						graph[i + 1][j].val = rand() % 100;
						graph[i + 1][j + 1].val = rand() % 100;

					}
				}
			}
			else if (i == size - 1) {
				if (j == 0) { //bot left corner
					if (graph[i - 1][j].val == -1 && graph[i][j + 1].val == -1 && graph[i - 1][j + 1].val == -1) {
						graph[i - 1][j].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i - 1][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
					}
				}
				else if (j == size - 1) { //bot right corner
					if (graph[i - 1][j].val == -1 && graph[i][j - 1].val == -1 && graph[i - 1][j - 1].val == -1) {
						graph[i - 1][j].val = rand() % 100;
						graph[i][j - 1].val = rand() % 100;
						graph[i - 1][j - 1].val = rand() % 100;
					}
				}
				else { //just on bot row
					if (graph[i][j - 1].val == -1 && graph[i][j + 1].val == -1 &&
						graph[i - 1][j - 1].val == -1 && graph[i - 1][j].val == -1 && graph[i - 1][j + 1].val == -1) {

						graph[i][j - 1].val = rand() % 100;
						graph[i][j + 1].val = rand() % 100;
						graph[i - 1][j - 1].val = rand() % 100;
						graph[i - 1][j].val = rand() % 100;
						graph[i - 1][j + 1].val = rand() % 100;

					}
				}
			}
			else {
				if (j == 0) { //mid row, left side of graph
					if (graph[i - 1][j].val == -1 && graph[i - 1][j + 1].val == -1 &&
						graph[i][j + 1].val == -1 &&
						graph[i + 1][j].val == -1 && graph[i + 1][j + 1].val == -1) {

						graph[i - 1][j].val = rand() % 100;
						graph[i - 1][j + 1].val = rand() % 100;

						graph[i][j + 1].val = rand() % 100;

						graph[i + 1][j].val = rand() % 100;
						graph[i + 1][j + 1].val = rand() % 100;
					}

				}
				else if (j == size - 1) { //mid row, right side of graph
					if (graph[i - 1][j - 1].val == -1 && graph[i - 1][j].val == -1 &&
						graph[i][j - 1].val == -1 &&
						graph[i + 1][j - 1].val == -1 && graph[i + 1][j].val == -1) {

						graph[i - 1][j - 1].val = rand() % 100;
						graph[i - 1][j].val = rand() % 100;

						graph[i][j - 1].val = rand() % 100;

						graph[i + 1][j - 1].val = rand() % 100;
						graph[i + 1][j].val = rand() % 100;
					}

				}
				else { //landlocked element
					if (graph[i - 1][j - 1].val == -1 && graph[i - 1][j].val == -1 && graph[i - 1][j + 1].val == -1 &&
						graph[i][j - 1].val == -1 && graph[i][j + 1].val == -1 &&
						graph[i + 1][j - 1].val == -1 && graph[i + 1][j].val == -1 && graph[i + 1][j + 1].val == -1) {

						graph[i - 1][j - 1].val = rand() % 100;
						graph[i - 1][j].val = rand() % 100;
						graph[i - 1][j + 1].val = rand() % 100;

						graph[i][j - 1].val = rand() % 100;
						graph[i][j + 1].val = rand() % 100;

						graph[i + 1][j - 1].val = rand() % 100;
						graph[i + 1][j].val = rand() % 100;
						graph[i + 1][j + 1].val = rand() % 100;
					}
				}
			}
		}
	}

	return graph;
}

void genGraph(vertex*** graph, int size) {
	*graph = (vertex**)malloc(sizeof(vertex*) * size);
	srand((unsigned int)time(NULL));

	int i = 0;
	for (; i < size; ++i) {
		vertex* a = (vertex*)malloc(sizeof(vertex) * size);
		int j = 0;
		for (; j < size; ++j) {
			int rando = rand() % 150; //this is 150 values (from 0 to 149),
			if (rando < 100) { //gives -1 more chance to appear
				a[j].val = rando;
			}
			else {
				a[j].val = -1; //too lazy to find how to include -1 in range when generating random number
			}
			a[j].vflag = 0;
		}

		(*graph)[i] = a;
	}

	*graph = checkGraph(*graph, size);
}


__device__ //altered to account for threads
Node bfsStart(vertex** graph, int size) {
	Node start;
	int i = 0;
	for(;i < size; ++i) {
		for (int j = 0; j < size; ++j) {
			__syncthreads();
			if ((*graph)[i*size + j].val != -1 && (*graph)[i*size+j].vflag == 0) { //thread has already occupied this start value
				atomicOr(&((*graph)[i*size + j].vflag), 1); //atomic Or sets flag, removes race condition
				int* holder = (int*)malloc(sizeof(*holder) * 2);
				*holder = i;
				*(holder + 1) = j;
				start.val = (*graph)[i*size + j].val;
				start.coord = holder;
				start.next = NULL;
				goto funcEnd; //TODO: changed this, now getting seg faults
			}
		}
	}
	funcEnd:
	return start;
}

__device__
void pushNode(vertex** graph, Node** qhead, int row, int col, int size, int** storageArray) {
	int val = (*graph)[row*size + col].val;
	**storageArray = (*graph)[row*size + col].val; //place val into storage array atomically
	*storageArray = *storageArray + 1;
	int* holder = (int*)malloc(sizeof(*holder) * 2);
	*holder = row;
	*(holder+1) = col;
	queuePush(qhead, val, holder);
}

//to be ran on each individual thread
__device__
void bfsCuda(vertex** graph, int size, Node currNode, Node** qhead, int** storageArray) {

	//queue used to store the value that you will move to next
	//the value that we move to is the "coord" member variable in Node struct

	//test adjacent nodes with start.coord value
	//NOTE: coord[0] is the row, coord[1] is the col

	Node next = currNode; //this will change in while loop

	while (1) {
		int deadEndFlag = 1; //flag for testing if all verticies around a vetex have been visited

		int r = *(next.coord);
		int c = *(next.coord+1);

		//dealing with 1d array here, so have to address based on size of row
		if(r - 1 >= 0) { //if not first row
			if((*graph)[(r-1)*size + c].val != -1 && (*graph)[(r-1)*size + c].vflag == 0) {
				(*graph)[(r - 1)*size + c].vflag = 1;
				pushNode(graph, qhead, r-1, c, size, storageArray);
				deadEndFlag = 0;
			}
			if(c - 1 >= 0) { //if not first row, not first col
				if((*graph)[(r - 1)*size + (c-1)].val != -1 && (*graph)[(r - 1)*size + (c - 1)].vflag == 0) {
					(*graph)[(r - 1)*size + (c - 1)].vflag = 1;
					pushNode(graph, qhead, r-1, c-1, size, storageArray);
					deadEndFlag = 0;
				}
				if((*graph)[(r)*size + (c - 1)].val != -1 && (*graph)[(r)*size + (c - 1)].vflag == 0) {
					(*graph)[(r)*size + (c - 1)].vflag = 1;
					pushNode(graph, qhead, r, c-1, size, storageArray);
					deadEndFlag = 0;
				}
			} 
			if (c + 1 < size) { //not first row, not last col
				if((*graph)[(r-1)*size + (c + 1)].val != -1 && (*graph)[(r - 1)*size + (c + 1)].vflag == 0) {
					(*graph)[(r - 1)*size + (c + 1)].vflag = 1;
					pushNode(graph, qhead, r-1, c+1, size, storageArray);
					deadEndFlag = 0;
				}
				if((*graph)[(r)*size + (c + 1)].val != -1 && (*graph)[(r)*size + (c + 1)].vflag == 0) {
					(*graph)[(r)*size + (c + 1)].vflag = 1;
					pushNode(graph, qhead, r, c+1, size, storageArray);
					deadEndFlag = 0;
				}
			}
		}
		if(r + 1 < size) { //not last row
			if((*graph)[(r+1)*size + (c)].val != -1 && (*graph)[(r + 1)*size + (c)].vflag == 0) {
				(*graph)[(r + 1)*size + (c)].vflag = 1;
				pushNode(graph, qhead, r+1, c, size, storageArray);
				deadEndFlag = 0;
			}
			if(c - 1 >= 0) { //not last row, not first col
				if((*graph)[(r)*size + (c-1)].val != -1 && (*graph)[(r)*size + (c - 1)].vflag == 0) {
					(*graph)[(r)*size + (c - 1)].vflag = 1;
					pushNode(graph, qhead, r, c-1, size, storageArray);
					deadEndFlag = 0;
				}
				if((*graph)[(r+1)*size + (c - 1)].val != -1 && (*graph)[(r + 1)*size + (c - 1)].vflag == 0) {
					(*graph)[(r + 1)*size + (c - 1)].vflag = 1;
					pushNode(graph, qhead, r+1, c-1, size, storageArray);
					deadEndFlag = 0;
				}
			}
			if(c + 1 < size) { //not last row, not last col
				if((*graph)[(r)*size + (c+1)].val != -1 && (*graph)[(r)*size + (c + 1)].vflag == 0) {
					(*graph)[(r)*size + (c + 1)].vflag = 1;
					pushNode(graph, qhead, r, c+1, size, storageArray);
					deadEndFlag = 0;
				}
				if ((*graph)[(r+1)*size + (c + 1)].val != -1 && (*graph)[(r + 1)*size + (c + 1)].vflag == 0) {
					(*graph)[(r + 1)*size + (c + 1)].vflag = 1;
					pushNode(graph, qhead, r+1, c+1, size, storageArray);
					deadEndFlag = 0;
				}
			}
		}

		//once each thread has marked surrounding nodes, then they select which node to go to next
		__syncthreads();


		//ACTUALLY, dont think this is a problen, if a thread flags a node, then the
		//thread that comes after it will see that it has already been flagged.
		//becaue the flag only needs to be set once, synchronization isnt really a problem as
		//far as the graph is concerned. just need to synch when storing found value into storage array
		//TODO: will need to have another check because a thread that has finished its check
		//does not see that its adjacent nodes have been

		//base case
		//base case check
		if(!deadEndFlag) { //need to do recursion
			next = queuePop(qhead);
		} else {
			if (*qhead != NULL) {
				next = queuePop(qhead);
			} else {
				break;
			}
		}
	}
}

__global__	void bfsCudaKernel(vertex* graph1d, int size, int* storageArray) {
	//int tIndex = threadIdx.x + blockIdx.x*blockDim.x;

	//each thread should have there own start place and queue
	//create queue
	Node* qheadbuff = nullptr;
	//initialize queue for each thread
	Node start = bfsStart(&graph1d, size);

	__syncthreads();

	//graph should be divided based on block indexing
	bfsCuda(&graph1d, size, start, &qheadbuff, &storageArray);



}


//fuction to change 2d array into 1d array
vertex* flatten(vertex** array2d, int size2d) {
	vertex* array1d = (vertex*)malloc(sizeof(vertex)*size2d*size2d);

	for (int i = 0; i < size2d; ++i) {
		for (int j = 0; j < size2d; ++j) {
			array1d[j + i*size2d] = array2d[i][j];
			//store the coordinates of the flattened array
			array1d[j + i*size2d].coordX = i;
			array1d[j + i*size2d].coordY = j;
		}
	}

	return array1d;
}


vertex** expand(vertex* array1d, int size2d) {
	vertex** array2d = (vertex**)malloc(sizeof(vertex*)*size2d*size2d);

	for (int i = 0; i < size2d; ++i) {
		for (int j = 0; j < size2d; ++j) {
			array2d[i][j] = array1d[j + i*size2d];
		}
	}

	return array2d;
}

int main(int argc, char* argv[]) {
	int graphSize = 10;
	int numBlocks = 1; //try to code for only threads for now
	int tPerBlock = 1;

	if(graphSize < 1 || numBlocks < 1 || tPerBlock < 1) {
		printf("invalid input!\n");
		return 0;
	}

	vertex** g;
	int valSize = sizeof(int) * (graphSize*graphSize);
	int* valStorage = (int*)malloc(valSize);
 	genGraph(&g, graphSize);

	vertex* g1d = flatten(g, graphSize);

	vertex* d_graph;
	int* d_valStorage; //store the result here (every thread does this)
	

	hipMalloc((void**)&d_graph, sizeof(vertex)*graphSize*graphSize);
	hipMalloc((void**)&d_valStorage, valSize);
	
	hipMemcpy(d_graph, g1d, sizeof(vertex)*graphSize*graphSize, hipMemcpyHostToDevice); //move graph to device

	bfsCudaKernel<<<1, tPerBlock>>>(d_graph, graphSize, d_valStorage);
	cudaCheckErrors("failed");

	hipMemcpy(g1d, d_graph, sizeof(vertex)*graphSize*graphSize, hipMemcpyDeviceToHost);
	hipMemcpy(valStorage, d_valStorage, valSize, hipMemcpyDeviceToHost);

	for (int k = 0; k < graphSize*graphSize; k++) {
		//printf("%d\n", g1d[k].val);
		printf("%d\n", g1d[k].vflag);
		//printf("%d\n", valStorage[k]);
	}

	g = expand(g1d, graphSize);
	printGraph(g, graphSize);

	int i = 0;
	for (; i < graphSize*graphSize; ++i) {
		if (valStorage[i] == -1) {
			break;
		}
		if (i % 10 != 9) {
			printf("%d, ", valStorage[i]);
		}
		else {
			printf("%d\n", valStorage[i]);
		}
	}
	printf("\n");

	hipFree(&d_graph);
	hipFree(&d_valStorage);


	free(g);
	free(g1d);
	free(valStorage);

	return 0;

}

//nvm i think this is covered by the checkGraph function
//TODO: need to divide the graph in the bfsCuda function such that each block is 
//allocated an even portion of the graph. the checkGraph() function does NOT
//hold if the graph is cut like this, so u want to add logic to account for this