#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "linklist.h"
#include "queue.h"
#include "graph.h"


__global__ //altered to account for threads
Node bfsStart(vertex*** graph, int size) {
	int i = 0;
	for(;i < size; ++i) {
		int j = 0;
		for(;j < size; ++j) {
			__synchthreads();
			if((*graph)[i][j].val != -1 && (*graph)[i][j].vflag == 0) { //thread has already occupied this start value
				atomicAdd((*graph)[i][j].vflag, 1); //atomic add removes race condition
				Node start;
				int* holder = malloc(sizeof(*holder) * 2);
				*holder = i;
				*(holder+1) = j;
				start.val = (*graph)[i][j].val;
				start.coord = holder;
				start.next = NULL;
				return start;
			}
		}
	}
}

__global__
void pushNode(vertex*** graph, Node** qhead, int row, int col, int** storageArray) {
	int val = (*graph)[row][col].val;
	atomicAdd(**storageArray, (*graph)[row][col].val); //place val into storage array atomically
	atomicAdd(*storageArray, 1);
	int* holder = malloc(sizeof(*holder) * 2);
	*holder = row;
	*(holder+1) = col;
	queuePush(qhead, val, holder);
}

//to be ran on each individual thread
__global__
void bfsIteration(vertex*** graph, int size, Node currNode, Node** qhead, int** storageArray) {

	//queue used to store the value that you will move to next
	//the value that we move to is the "coord" member variable in Node struct

	//test adjacent nodes with start.coord value
	//NOTE: coord[0] is the row, coord[1] is the col

	Node next = currNode; //this will change in while loop

	while (1) {
		int deadEndFlag = 1; //flag for testing if all verticies around a vetex have been visited

		int r = *(next.coord);
		int c = *(next.coord+1);

		//linklistTraverse(*qhead);

		// printf("r is %d\n", r);
		// printf("c is %d\n", c);

		// printf("%d\n", (*graph)[r][c].val);

		if(r - 1 >= 0) { //if not first row
			if((*graph)[r-1][c].val != -1 && (*graph)[r-1][c].vflag == 0) {
				(*graph)[r-1][c].vflag = 1;
				pushNode(graph, qhead, r-1, c, storageArray);
				deadEndFlag = 0;
			}
			if(c - 1 >= 0) { //if not first row, not first col
				if((*graph)[r-1][c-1].val != -1 && (*graph)[r-1][c-1].vflag == 0) {
					(*graph)[r-1][c-1].vflag = 1;
					pushNode(graph, qhead, r-1, c-1, storageArray);
					deadEndFlag = 0;
				}
				if((*graph)[r][c-1].val != -1 && (*graph)[r][c-1].vflag == 0) {
					(*graph)[r][c-1].vflag = 1;
					pushNode(graph, qhead, r, c-1, storageArray);
					deadEndFlag = 0;
				}
			} 
			if (c + 1 < size) { //not first row, not last col
				if((*graph)[r-1][c+1].val != -1 && (*graph)[r-1][c+1].vflag == 0) {
					(*graph)[r-1][c+1].vflag = 1;
					pushNode(graph, qhead, r-1, c+1, storageArray);
					deadEndFlag = 0;
				}
				if((*graph)[r][c+1].val != -1 && (*graph)[r][c+1].vflag == 0) {
					(*graph)[r][c+1].vflag = 1;
					pushNode(graph, qhead, r, c+1, storageArray);
					deadEndFlag = 0;
				}
			}
		}
		if(r + 1 < size) { //not last row
			if((*graph)[r+1][c].val != -1 && (*graph)[r+1][c].vflag == 0) {
				(*graph)[r+1][c].vflag = 1;
				pushNode(graph, qhead, r+1, c, storageArray);
				deadEndFlag = 0;
			}
			if(c - 1 >= 0) { //not last row, not first col
				if((*graph)[r][c-1].val != -1 && (*graph)[r][c-1].vflag == 0) {
					(*graph)[r][c-1].vflag = 1;
					pushNode(graph, qhead, r, c-1, storageArray);
					deadEndFlag = 0;
				}
				if((*graph)[r+1][c-1].val != -1 && (*graph)[r+1][c-1].vflag == 0) {
					(*graph)[r+1][c-1].vflag = 1;
					pushNode(graph, qhead, r+1, c-1, storageArray);
					deadEndFlag = 0;
				}
			}
			if(c + 1 < size) { //not last row, not last col
				if((*graph)[r][c+1].val != -1 && (*graph)[r][c+1].vflag == 0) {
					(*graph)[r][c+1].vflag = 1;
					pushNode(graph, qhead, r, c+1, storageArray);
					deadEndFlag = 0;
				}
				if ((*graph)[r+1][c+1].val != -1 && (*graph)[r+1][c+1].vflag == 0) {
					(*graph)[r+1][c+1].vflag = 1;
					pushNode(graph, qhead, r+1, c+1, storageArray);
					deadEndFlag = 0;
				}
			}
		}

		//once each thread has marked surrounding nodes, then they select which node to go to next
		__synchthreads();


		//ACTUALLY, dont think this is a problen, if a thread flags a node, then the
		//thread that comes after it will see that it has already been flagged.
		//becaue the flag only needs to be set once, synchronization isnt really a problem as
		//far as the graph is concerned. just need to synch when storing found value into storage array
		//TODO: will need to have another check because a thread that has finished its check
		//does not see that its adjacent nodes have been

		//base case
		//base case check
		if(!deadEndFlag) { //need to do recursion
			next = queuePop(qhead);
		} else {
			if (*qhead != NULL) {
				next = queuePop(qhead);
			} else {
				break;
			}
		}

	}
}

__global__
void bfsCuda(vertex*** graph, vertex*** splitGraph, int size, Node** qheadbuff, int** storageArray) {
	int tIndex = threadIdx.x + blockIdx.x*blockDim.x;

	//each thread should have there own start place and queue
	//create queue

	sgSize = size/gridDim.x;

	__shared__ vertex** sGraph = splitGraph;

	//split graph for each block
	int i = 0;
	for(;i < sgSize; ++i) {
		sGraph[i] = graph[i + blockIdx.x];
	}

	//initialize queue for each thread
	qheadbuff[tIndex] = bfsStart(sGraph, sgSize);

	__synchthreads();

	//graph should be divided based on block indexing
	bfsIteration(splitGraph, sgSize, start, qheadBuff[tIndex], storageArray);
}

int main(int argc, char* argv[]) {
	int graphSize = atoi(argv[1]);
	int numBlocks = atoi(argv[2]);
	int tPerBlock = atoi(argv[3]);

	if(graphSize < 1 || numBlocks < 1 || numThreads < 1) {
		printf("invalid input!\n");
		return 0;
	}

	vertex** g;
 	genGraph(&g, graphSize);

	vertex** d_graph;
	vertex** d_sGraph; //split the graph for each block
	int* valStorage; //store the result here (every thread does this)
	node** qheadHolder = sizeof(*Node)*(tPerBlock*numBlocks);
	node** d_qheadHolder; //stores each queue for all the threads

	int i = 0;
	for(;i < tPerBlock*numBlocks; ++i) {
		Node* qhead = NULL;
		qheadHolder[i] = qhead;
	}

	hipMalloc(d_graph, sizeof(g));
	hipMalloc(d_sGraph, sizeof(graph)/numBlocks); //will be populated in function
	hipMalloc(valStorage, sizeof(*valStorage) * (graphSize*graphSize));
	hipMalloc(d_qheadHolder, sizeof(*Node)*(tPerBlock*numBlocks));
	hipMemcpy(d_graph, g, sizeof(g), hipMemcpyHostToDevice); //move graph to device
	hipMemcpy(d_qheadHolder, qheadHolder, sizeof(qheadHolder), hipMemcpyHostToDevice); //move all heads to device

	free(qheadHolder);
	free(g);

	bfsCuda<<<numBlocks, numThreads>>>(&d_graph, &d_sGraph, graphSize, d_qheadHolder, &valStorage);

	hipFree(d_graph);
	hipFree(d_sGraph);
	hipFree(valStorage);
	hipFree(d_qheadHolder);


	return 0;

}

//nvm i think this is covered by the checkGraph function
//TODO: need to divide the graph in the bfsCuda function such that each block is 
//allocated an even portion of the graph. the checkGraph() function does NOT
//hold if the graph is cut like this, so u want to add logic to account for this