#include "hip/hip_runtime.h"
//Breadth First Search CUDA implementation
//Written By: Cedric Blake

//Compiled and ran using Microsoft Visual Studio 2015


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "sm_20_atomic_functions.h"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } else { \
			printf("success\n"); \
		} \
    } while (0)

#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
int atomicOr(int*, int);
int atomicExch(int*, int);
int atomicAdd(int*, int);
#endif

typedef struct vertex {
	int val;
	int vflag;
	int coordX, coordY;
} vertex;

typedef struct Node {
	int val;
	int* coord; //used to find value on graph array
	struct Node* next;
} Node;


//graph.c functions

void printGraph(vertex** graph, int size) {

	printf("\nGraph Values\n\n");
	int i = 0;
	for (; i < size; ++i) {
		int j = 0;
		for (; j < size; ++j) {
			if (graph[i][j].val != -1) {
				if (graph[i][j].val < 10) {
					printf("%d  ", graph[i][j].val);
				}
				else {
					printf("%d ", graph[i][j].val);
				}
			}
			else {
				printf("   ");
			}
		}
		printf("\n");
	}

	printf("\n");

	printf("Visited Flag Values\n\n");
	for (i = 0; i < size; ++i) {
		int j = 0;
		for (; j < size; ++j) {
			if (graph[i][j].val != -1) {
				printf("%d ", graph[i][j].vflag);
			}
			else {
				printf("  ");
			}
		}
		printf("\n");
	}
}

vertex** checkGraph(vertex** g, int size) { //for simplicity, change every value around isolated value to (not -1)
	vertex** graph;
	graph = g;
	int i = 0;
	for (; i < size; ++i) {
		int j = 0;
		for (; j < size; ++j) {
			if (i == 0) { //top row
				if (j == 0) { //top left corner
					if (graph[i + 1][j].val == -1 && graph[i][j + 1].val == -1 && graph[i + 1][j + 1].val == -1) {
						graph[i + 1][j].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i + 1][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
					}
				}
				else if (j == size - 1) { //top right corner
					if (graph[i + 1][j].val == -1 && graph[i][j - 1].val == -1 && graph[i + 1][j - 1].val == -1) {
						graph[i + 1][j].val = rand() % 100;
						graph[i][j - 1].val = rand() % 100;
						graph[i + 1][j - 1].val = rand() % 100;
					}
				}
				else { //just on top row
					if (graph[i][j - 1].val == -1 && graph[i][j + 1].val == -1 &&
						graph[i + 1][j - 1].val == -1 && graph[i + 1][j].val == -1 && graph[i + 1][j + 1].val == -1) {

						graph[i][j - 1].val = rand() % 100;
						graph[i][j + 1].val = rand() % 100;
						graph[i + 1][j - 1].val = rand() % 100;
						graph[i + 1][j].val = rand() % 100;
						graph[i + 1][j + 1].val = rand() % 100;

					}
				}
			}
			else if (i == size - 1) {
				if (j == 0) { //bot left corner
					if (graph[i - 1][j].val == -1 && graph[i][j + 1].val == -1 && graph[i - 1][j + 1].val == -1) {
						graph[i - 1][j].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
						graph[i - 1][j + 1].val = rand() % 100; //this is 100 (from 0 to 99)
					}
				}
				else if (j == size - 1) { //bot right corner
					if (graph[i - 1][j].val == -1 && graph[i][j - 1].val == -1 && graph[i - 1][j - 1].val == -1) {
						graph[i - 1][j].val = rand() % 100;
						graph[i][j - 1].val = rand() % 100;
						graph[i - 1][j - 1].val = rand() % 100;
					}
				}
				else { //just on bot row
					if (graph[i][j - 1].val == -1 && graph[i][j + 1].val == -1 &&
						graph[i - 1][j - 1].val == -1 && graph[i - 1][j].val == -1 && graph[i - 1][j + 1].val == -1) {

						graph[i][j - 1].val = rand() % 100;
						graph[i][j + 1].val = rand() % 100;
						graph[i - 1][j - 1].val = rand() % 100;
						graph[i - 1][j].val = rand() % 100;
						graph[i - 1][j + 1].val = rand() % 100;

					}
				}
			}
			else {
				if (j == 0) { //mid row, left side of graph
					if (graph[i - 1][j].val == -1 && graph[i - 1][j + 1].val == -1 &&
						graph[i][j + 1].val == -1 &&
						graph[i + 1][j].val == -1 && graph[i + 1][j + 1].val == -1) {

						graph[i - 1][j].val = rand() % 100;
						graph[i - 1][j + 1].val = rand() % 100;

						graph[i][j + 1].val = rand() % 100;

						graph[i + 1][j].val = rand() % 100;
						graph[i + 1][j + 1].val = rand() % 100;
					}

				}
				else if (j == size - 1) { //mid row, right side of graph
					if (graph[i - 1][j - 1].val == -1 && graph[i - 1][j].val == -1 &&
						graph[i][j - 1].val == -1 &&
						graph[i + 1][j - 1].val == -1 && graph[i + 1][j].val == -1) {

						graph[i - 1][j - 1].val = rand() % 100;
						graph[i - 1][j].val = rand() % 100;

						graph[i][j - 1].val = rand() % 100;

						graph[i + 1][j - 1].val = rand() % 100;
						graph[i + 1][j].val = rand() % 100;
					}

				}
				else { //landlocked element
					if (graph[i - 1][j - 1].val == -1 && graph[i - 1][j].val == -1 && graph[i - 1][j + 1].val == -1 &&
						graph[i][j - 1].val == -1 && graph[i][j + 1].val == -1 &&
						graph[i + 1][j - 1].val == -1 && graph[i + 1][j].val == -1 && graph[i + 1][j + 1].val == -1) {

						graph[i - 1][j - 1].val = rand() % 100;
						graph[i - 1][j].val = rand() % 100;
						graph[i - 1][j + 1].val = rand() % 100;

						graph[i][j - 1].val = rand() % 100;
						graph[i][j + 1].val = rand() % 100;

						graph[i + 1][j - 1].val = rand() % 100;
						graph[i + 1][j].val = rand() % 100;
						graph[i + 1][j + 1].val = rand() % 100;
					}
				}
			}
		}
	}

	return graph;
}

void genGraph(vertex*** graph, int size) {
	*graph = (vertex**)malloc(sizeof(vertex*) * size);
	srand((unsigned int)time(NULL));

	int i = 0;
	for (; i < size; ++i) {
		vertex* a = (vertex*)malloc(sizeof(vertex) * size);
		int j = 0;
		for (; j < size; ++j) {
			int rando = rand() % 150; //this is 150 values (from 0 to 149),
			if (rando < 100) { //gives -1 more chance to appear
				a[j].val = rando;
			}
			else {
				a[j].val = -1; //too lazy to find how to include -1 in range when generating random number
			}
			a[j].vflag = 0;
		}

		(*graph)[i] = a;
	}

	*graph = checkGraph(*graph, size);
}

void destroyGraph(vertex** graph, int size) {
	int i = 0;
	for(; i < size; ++i) {
		free(graph[i]);
	}

	free(graph);
	printf("graph destroyed!\n");
}

void writeGraph(vertex** graph, char* fileName, long int size) {
	FILE* f = fopen(fileName, "wb");

	int* array1d = (int*)malloc(sizeof(int)* size * size);

	//flatten graph
	int i = 0;
	for (; i < size; ++i) {
		int j = 0;
		for (; j < size; ++j) {
			array1d[j + i*size] = graph[i][j].val;
		}
	}


	printf("Writing Array\n");
	i = 0;
	fwrite(array1d, sizeof(int), size*size, f);
	fclose(f);
	free(array1d);
}

vertex** readGraph(char * filename, long int size) {
	int* array1d = (int*)malloc(sizeof(int) * size * size);

	FILE *f = fopen(filename, "rb");
	if (f == NULL) {
		printf("Error reading File\n");
		return NULL;
	}
	printf("Reading Array\n");
	fread(array1d, sizeof(int), size*size, f);

	vertex** graph = (vertex**)malloc(sizeof(vertex*) * size * size);
	//expand graph
	int i = 0;
	for (; i < size; ++i) {
		graph[i] = (vertex*)malloc(sizeof(vertex)*size);
		int j = 0;
		for (; j < size; ++j) {
			graph[i][j].val = array1d[j + i*size];
			graph[i][j].vflag = 0;
		}
	}

	free(array1d);

	return graph;
}

//to be ran on each individual thread
__device__ void bfsCuda(vertex* graph, int size, Node currNode) {

	//queue used to store the value that you will move to next
	//the value that we move to is the "coord" member variable in Node struct

	//test adjacent nodes with start.coord value
	//NOTE: coord[0] is the row, coord[1] is the col

	Node next = currNode; //this will change in while loop
	int r = *(next.coord);
	int c = *(next.coord + 1);

	if (graph[r*size + c].val != -1) {

		//dealing with 1d array here, so have to address based on size of row
		if (r - 1 >= 0) { //if not first row
			if (graph[(r - 1)*size + c].val != -1 && graph[(r - 1)*size + c].vflag == 0) {
				graph[(r - 1)*size + c].vflag = 1;
			}
			if (c - 1 >= 0) { //if not first row, not first col
				if (graph[(r - 1)*size + (c - 1)].val != -1 && graph[(r - 1)*size + (c - 1)].vflag == 0) {
					graph[(r - 1)*size + (c - 1)].vflag = 1;
				}
				if (graph[(r)*size + (c - 1)].val != -1 && graph[(r)*size + (c - 1)].vflag == 0) {
					graph[(r)*size + (c - 1)].vflag = 1;
				}
			}
			if (c + 1 < size) { //not first row, not last col
				if (graph[(r - 1)*size + (c + 1)].val != -1 && graph[(r - 1)*size + (c + 1)].vflag == 0) {
					graph[(r - 1)*size + (c + 1)].vflag = 1;
				}
				if (graph[(r)*size + (c + 1)].val != -1 && graph[(r)*size + (c + 1)].vflag == 0) {
					graph[(r)*size + (c + 1)].vflag = 1;
				}
			}
		}
		if (r + 1 < size) { //not last row
			if (graph[(r + 1)*size + (c)].val != -1 && graph[(r + 1)*size + (c)].vflag == 0) {
				graph[(r + 1)*size + (c)].vflag = 1;
			}
			if (c - 1 >= 0) { //not last row, not first col
				if (graph[(r)*size + (c - 1)].val != -1 && graph[(r)*size + (c - 1)].vflag == 0) {
					graph[(r)*size + (c - 1)].vflag = 1;
				}
				if (graph[(r + 1)*size + (c - 1)].val != -1 && graph[(r + 1)*size + (c - 1)].vflag == 0) {
					graph[(r + 1)*size + (c - 1)].vflag = 1;
				}
			}
			if (c + 1 < size) { //not last row, not last col
				if (graph[(r)*size + (c + 1)].val != -1 && graph[(r)*size + (c + 1)].vflag == 0) {
					graph[(r)*size + (c + 1)].vflag = 1;
				}
				if (graph[(r + 1)*size + (c + 1)].val != -1 && graph[(r + 1)*size + (c + 1)].vflag == 0) {
					graph[(r + 1)*size + (c + 1)].vflag = 1;
				}
			}
		}
	}
}

__global__	void bfsCudaKernel(vertex* graph1d, int size) {
	int tIndex = threadIdx.x + blockIdx.x*blockDim.x;

	//each thread should have there own start place and queue
	//create queue

	//graph should be divided based on block indexing
	//bfsCuda(graph1d, size, startHolder[tIndex], qheadbuff);

	Node start;
	int* holder = (int*)malloc(sizeof(*holder) * 2);
	*holder = graph1d[tIndex].coordX;
	*(holder + 1) = graph1d[tIndex].coordY;
	start.val = graph1d[tIndex].val;
	start.coord = holder;
	start.next = nullptr;

	for (int i = tIndex; i < size*size; ++i) {
		bfsCuda(graph1d, size, start);
	}

	free(holder);
	__syncthreads();

}


//fuction to change 2d array into 1d array
void flatten(vertex** array2d, vertex** array1d, int size2d) {

	for (int i = 0; i < size2d; ++i) {
		for (int j = 0; j < size2d; ++j) {
			(*array1d)[j + i*size2d] = array2d[i][j];
			//store the coordinates of the flattened array
			(*array1d)[j + i*size2d].coordX = i;
			(*array1d)[j + i*size2d].coordY = j;
		}
	}
}

void expand(vertex*** graph, vertex* array1d, int size2d) {

	for (int i = 0; i < size2d; ++i) {
		for (int j = 0; j < size2d; ++j) {
			(*graph)[i][j].vflag = array1d[j + i*size2d].vflag;
		}
	}
}

int main(int argc, char* argv[]) {
	int graphSize = 100;
	//int numBlocks = 100;
	int tPerBlock = 100;

	srand((unsigned int)time(NULL));

	char* fileName = "graphsave.txt";
	int genFlag = 0;
	//create graph
	vertex** g = readGraph(fileName, graphSize);
	if (g == NULL) {
		genGraph(&g, graphSize);
		genFlag = 1;
	}
	
	vertex* g1d = (vertex*)malloc(sizeof(*g1d)*graphSize*graphSize);
	flatten(g, &g1d, graphSize);

	
	vertex* d_graph;
	
	hipSetDevice(0);
	
	hipMalloc((void**)&d_graph, sizeof(vertex)*graphSize*graphSize);

	
	hipMemcpy(d_graph, g1d, sizeof(vertex)*graphSize*graphSize, hipMemcpyHostToDevice); //move graph to device
	
	clock_t begin = clock();
	bfsCudaKernel<<<(graphSize*graphSize)/tPerBlock, tPerBlock>>>(d_graph, graphSize);
	clock_t end = clock();

	float timeTaken = (float)((end - begin) / (float)CLOCKS_PER_SEC);

	hipMemcpy(g1d, d_graph, sizeof(vertex)*graphSize*graphSize, hipMemcpyDeviceToHost);
	cudaCheckErrors("failed");

	expand(&g, g1d, graphSize);
	
	/*
	for (int k = 0; k < graphSize*graphSize; k++) {
		//printf("%d\n", g1d[k].val);
		printf("%d, ", g1d[k].val);
		//printf("%d\n", valStorage[k]);
	}
	*/

	hipFree(&d_graph);

	printGraph(g, graphSize);

	printf("\n");
	printf("time taken: %f\n", timeTaken);

	if (genFlag) { //then we generated a graph that needs to be written
		writeGraph(g, fileName, graphSize); // save graph to file
	}

	destroyGraph(g, graphSize);
	free(g1d);
	return 0;

}

//nvm i think this is covered by the checkGraph function
//TODO: need to divide the graph in the bfsCuda function such that each block is 
//allocated an even portion of the graph. the checkGraph() function does NOT
//hold if the graph is cut like this, so u want to add logic to account for this