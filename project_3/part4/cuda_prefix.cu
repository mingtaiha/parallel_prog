#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <cmath>
#include <hip/device_functions.h>

#define N 1048576
#define THREADS_PER_BLOCK 1024

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } else { \
			printf("success\n"); \
		} \
    } while (0)


#pragma once
#ifdef __INTELLISENSE__
void __syncthreads(); //Just so sync threads doesnt get underlined in red
int atomicAdd(int* address, int val);
#endif

typedef struct Node {
	int sum;
	int fromLeft;
}Node;


void randomGen(int *input, int size) {
	srand(time(NULL));
	for (int i = 0; i < size; i++) {
		input[i] = rand() % 100;
	}
}

__device__ void downPass(Node *nodeArray, int count) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	//set root node
	if (index%N == 0) {
		nodeArray[0].fromLeft = 0;
	}

	if (index%(N) == 0) {
		nodeArray[1].fromLeft = nodeArray[0].fromLeft;
		nodeArray[2].fromLeft = nodeArray[0].fromLeft + nodeArray[1].sum;
		
	}

	__syncthreads();

	int divisor = N / 2;
	for (int i = 1; i < count; i++) {
		if (index%divisor == 0) {
			//int random = i - 1;
			//int leftIndex = power(2, i) + index / divisor + 2 * random + 1;
			int leftIndex = 2 * i + 1;
			nodeArray[leftIndex].fromLeft = nodeArray[leftIndex / 2].fromLeft;//left child
			nodeArray[leftIndex + 1].fromLeft = nodeArray[leftIndex / 2].fromLeft + nodeArray[leftIndex].sum;
			divisor /= 2;
		}
		__syncthreads();
	}
}

//Tree builds!!!!
__device__ void buildTree(int *input, Node *nodeArray, int *sum, int count) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	nodeArray[index + (N-1)].sum = input[index]; //Save all the leaf nodes
	int divisor = 2;
	for (int i = 0; i < count; i++) { //Generate all of the parent nodes
		if (index % divisor == 0) {
			nodeArray[(index+(N-1))/divisor].sum = nodeArray[(index+(N-1))/(divisor/2)].sum + nodeArray[(index + N)/(divisor/2)].sum;
			divisor *= 2;
		}
		__syncthreads();
	}

}

__global__ void find_repeats(int *a, int *b, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) {
		if (a[index] == a[index + 1]) {
			b[index] = 1;
		}
		else {
			b[index] = 0;
		}
	}
}

__global__ void exclusive_scan(int *input, int *output, Node *nodeArray, int *totalSum, int iterations) {
	//int index = threadIdx.x + blockIdx.x * blockDim.x;
	buildTree(input, nodeArray, totalSum, iterations);
	__syncthreads();
	downPass(nodeArray, iterations);
	

}

//In order to implement this we need a working version of exclusive scan
__global__ void index_repeats(int *b, int *c, int *B) {

}

int main() {

	int *a, *b, *c; //host ABC
	int *dev_a, *dev_b, *dev_A;
	int *totalSum;
	int sum = 0;
	//int *dev_c, *dev_B;
	int size = N * sizeof(int);

	Node *nodeArray, *dev_nodeArray;
	int nodeArraySize = (2 * N - 1) * sizeof(Node);

	//cudamalloc a, b, and c on device memory

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_A, size);
	hipMalloc((void**)&totalSum, sizeof(int));
	hipMalloc((void**)&dev_nodeArray, nodeArraySize);
	//cudaMalloc((void**)&dev_c, size);
	//cudaMalloc((void**)&dev_B, size);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);
	nodeArray = (Node *)malloc(nodeArraySize);

	//Make random array
	randomGen(a, N);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	//This works properly
	find_repeats <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, N);


	//Do exclusive scan on dev_b, store in dev_c
	//exclusive_scan <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_b, dev_c);

	int iterations = log(N) / log(2); //Cannot be called from the device
	exclusive_scan <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_A, dev_nodeArray, totalSum, iterations);
	//Create B and C with some cuda operations on dev_c
	//need to implement exclusive scan on dev_b in order to do index repeats
	//index_repeats <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_b, dev_c, dev_B);

	hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);
	//cudaCheckErrors("failed");
	//cudaMemcpy(a, dev_A, size, cudaMemcpyDeviceToHost);
	hipMemcpy(&sum, totalSum, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(nodeArray, dev_nodeArray, nodeArraySize, hipMemcpyDeviceToHost);
	

	//remove repeats cannot be done in parallel
	int j = 0;
	for (int i = 0; i < N; i++) {
		if (b[i] == 0) {
			c[j] = a[i];
			j++;
		}
	}


	printf("Last element of find_repeats: %d\n", c[N-j]); //replace

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_A);
	hipFree(totalSum);
	hipFree(dev_nodeArray);
	//cudaFree(dev_c);
	free(a); free(b); free(c); free(nodeArray);
	return 0;

}
