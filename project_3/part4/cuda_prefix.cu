#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <cmath>
#include <hip/device_functions.h>

#define N 8
#define THREADS_PER_BLOCK 2

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } else { \
			printf("success\n"); \
		} \
    } while (0)


#pragma once
#ifdef __INTELLISENSE__
void __syncthreads(); //Just so sync threads doesnt get underlined in red
int atomicAdd(int* address, int val);
#endif

typedef struct Node {
	int sum;
	int fromLeft;
}Node;


void randomGen(int *input, int size) {
	srand(time(NULL));
	for (int i = 0; i < size; i++) {
		input[i] = rand() % 10;
	}
}


//This is working properly
__device__ void parallelAdd(int *input, int *sum, int count) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int divisor = 2;
	for (int i = 0; i < count; i++) {
		if (index % divisor == 0) {
			input[index] = input[index] + input[index + divisor/2];
			divisor *= 2;
		}
		__syncthreads();
	}

	if (index == 0) { //Initializing the Node
		*sum = input[0] - input[N - 1]; //exclusive scan does not include the last value
	}
	
}

__device__ int power(int a, int b) {
	for (int i = 0; i < b; i++) {
		a *= a;
	}
	return a;
}

__device__ void downPass(Node *nodeArray, int count) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	//set root node
	if (index%N == 0) {
		nodeArray[0].fromLeft = 0;
	}

	if (index%(N) == 0) {
		nodeArray[1].fromLeft = nodeArray[0].fromLeft;
		nodeArray[2].fromLeft = nodeArray[0].fromLeft + nodeArray[1].sum;
		
	}

	__syncthreads();


	if (index == 0) {
		nodeArray[3].fromLeft = nodeArray[1].fromLeft;
		nodeArray[4].fromLeft = nodeArray[1].fromLeft + nodeArray[3].sum;
	}
	if (index == 4) {
		nodeArray[5].fromLeft = nodeArray[2].fromLeft;
		nodeArray[6].fromLeft = nodeArray[2].fromLeft + nodeArray[5].sum;
	}
	
	__syncthreads();

	/*int divisor = N / 2;
	for (int i = 1; i < count; i++) {
		if (index%divisor == 0) {
			//int random = i - 1;
			//int leftIndex = power(2, i) + index / divisor + 2 * random + 1;
			int leftIndex = 2 * i + 1;
			nodeArray[leftIndex].fromLeft = nodeArray[leftIndex / 2].fromLeft;//left child
			nodeArray[leftIndex + 1].fromLeft = nodeArray[leftIndex / 2].fromLeft + nodeArray[leftIndex].sum;
			divisor /= 2;
		}
		__syncthreads();
	}*/
}

//Tree builds!!!!
__device__ void buildTree(int *input, Node *nodeArray, int *sum, int count) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	nodeArray[index + (N-1)].sum = input[index]; //Save all the leaf nodes
	int divisor = 2;
	for (int i = 0; i < count; i++) {
		if (index % divisor == 0) {
			nodeArray[(index+(N-1))/divisor].sum = nodeArray[(index+(N-1))/(divisor/2)].sum + nodeArray[(index + N)/(divisor/2)].sum;
			divisor *= 2;
		}
		__syncthreads();
	}

}

__global__ void find_repeats(int *a, int *b, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) {
		if (a[index] == a[index + 1]) {
			b[index] = 1;
		}
		else {
			b[index] = 0;
		}
	}
}

__global__ void exclusive_scan(int *input, int *output, Node *nodeArray, int *totalSum, int iterations) {
	//int index = threadIdx.x + blockIdx.x * blockDim.x;
	buildTree(input, nodeArray, totalSum, iterations);
	__syncthreads();
	downPass(nodeArray, iterations);
	

}

//In order to implement this we need a working version of exclusive scan
__global__ void index_repeats(int *b, int *c, int *B) {

}

int main() {

	int *a, *b, *c; //host ABC
	int *dev_a, *dev_b, *dev_A;
	int *totalSum;
	int sum = 0;
	//int *dev_c, *dev_B;
	int size = N * sizeof(int);

	Node *nodeArray, *dev_nodeArray;
	int nodeArraySize = (2 * N - 1) * sizeof(Node);

	//cudamalloc a, b, and c on device memory

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_A, size);
	hipMalloc((void**)&totalSum, sizeof(int));
	hipMalloc((void**)&dev_nodeArray, nodeArraySize);
	//cudaMalloc((void**)&dev_c, size);
	//cudaMalloc((void**)&dev_B, size);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);
	nodeArray = (Node *)malloc(nodeArraySize);

	//Make random array
	randomGen(a, N);

	for (int i = 0; i < N; i++) {
		printf("%d ", a[i]);
	}
	printf("\n");

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	//This works properly
	find_repeats <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, N);


	//Do exclusive scan on dev_b, store in dev_c
	//exclusive_scan <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_b, dev_c);

	int iterations = log(N) / log(2); //Cannot be called from the device
	exclusive_scan <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_A, dev_nodeArray, totalSum, iterations);
	//Create B and C with some cuda operations on dev_c
	//need to implement exclusive scan on dev_b in order to do index repeats
	//index_repeats <<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_b, dev_c, dev_B);

	hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);
	//cudaCheckErrors("failed");
	//cudaMemcpy(a, dev_A, size, cudaMemcpyDeviceToHost);
	hipMemcpy(&sum, totalSum, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(nodeArray, dev_nodeArray, nodeArraySize, hipMemcpyDeviceToHost);
	

	//remove repeats cannot be done in parallel
	int j = 0;
	for (int i = 0; i < N; i++) {
		if (b[i] == 0) {
			c[j] = a[i];
			j++;
		}
	}

	//print out the node array
	for (int i = 0; i < (2*N-1); i++) {
		printf("%d ", nodeArray[i].sum);
	}
	printf(" \n");

	//print out the node array
	for (int i = 0; i < (2 * N - 1); i++) {
		printf("%d ", nodeArray[i].fromLeft);
	}
	printf(" \n");



	//print out the c array
	for (int i = 0; i < N; i++) {
		printf("%d ", c[i]);
	}
	printf("\n");

	printf("Last element of find_repeats: %d\n", 0); //replace

	hipFree(dev_a);
	hipFree(dev_b);
	//cudaFree(dev_c);
	free(a); free(b); free(c);
	return 0;

}
