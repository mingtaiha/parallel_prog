#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

//Cuda implementation of prefix finder
//Structure from NVIDIA tutorial slides
#define N_SIZE 1000000
#define THREADS_PER_BLOCK 512
#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

__global__ void find_repeats(int *dev_a, int *dev_b, int n) {


}
__global__ void exclusive_scan(int *dev_b, int *dev_c, int n) {
	extern __shared__ float temp[];  // allocated on invocation  
	int thid = threadIdx.x;
	int offset = 1; 
	int ai = thid;
	int bi = thid + (n / 2);
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
		temp[ai + bankOffsetA] = dev_b[ai];
		temp[bi + bankOffsetB] = dev_b[bi];

	for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree  
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			temp[bi] += temp[ai];
		}
		offset *= 2;

		if (thid == 0) { temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }
		for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
		{
			offset >>= 1;
			__syncthreads();
			if (thid < d)
			{
				int ai = offset*(2 * thid + 1) - 1;
				int bi = offset*(2 * thid + 2) - 1;
				ai += CONFLICT_FREE_OFFSET(ai);
				bi += CONFLICT_FREE_OFFSET(bi);
				float t = temp[ai];
				temp[ai] = temp[bi];
				temp[bi] += t;
			}
		}
		__syncthreads();
		dev_c[ai] = temp[ai + bankOffsetA];
		dev_c[bi] = temp[bi + bankOffsetB];
	}

}
__global__ void index_repeats(int *dev_b, int *dev_c, int *dev_B, int n) {


}
__global__ void remove_repeats(int *dev_a, int *dev_B, int*dev_C, int n) {


}

main() {

	int *a, *b, *c; //host ABC
	int *dev_a, *dev_b, *dev_c, *dev_B, *dev_C;
	int size = N_SIZE * sizeof(int);
	int i;

	//cudamalloc a, b, and c on device memory

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	hipMalloc((void**)&dev_B, size);
	hipMalloc((void**)&dev_C, size);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	//Make random array
	for (i = 0; i < N_SIZE; i++) {
		a[i] = (int)(rand() % 100);
	}

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	//Do find_repeats, store in dev_b
	find_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, N_SIZE);


	//Do exclusive scan on dev_b, store in dev_c
	exclusive_scan <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_b, dev_c, N_SIZE);

	//Create B and C with some cuda operations on dev_c

	index_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_b, dev_c, dev_B, N_SIZE);

	remove_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_B, dev_C, N_SIZE);

	hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	printf("Last element of find_repeats: %d\n", 0); //replace

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(a); free(b); free(c);
	return 0;

}



