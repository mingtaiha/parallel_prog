#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "matrix_gen.h"

//Cuda implementation of prefix finder
//Structure from NVIDIA tutorial slides
#define N_SIZE 32
#define THREADS_PER_BLOCK 8
#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

__global__ void find_repeats(float *dev_a, float *dev_b, int n) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) {
		if (dev_a[index] == dev_a[index + 1]) {
			dev_b[index] = 1;
			printf("Found Repeat\t");
		}
		else {
			dev_b[index] = 0;
		}
	}
}
__global__ void addToAll(float *dev_c, float toAdd) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	dev_c[index] += toAdd;
}

__global__ void exclusive_scan(float *dev_b, float *dev_c, int n) {
	__shared__ float temp[THREADS_PER_BLOCK];  // allocated on invocation  
	int thid = threadIdx.x + blockIdx.x * blockDim.x;
	int block_offset = blockIdx.x * blockDim.x;
	int offset = 1; 
	int ai = 2*thid;
	int bi = 2*thid + 1;// +(n / 2);
	//int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	//int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
	//ai += bankOffsetA;
	//bi += bankOffsetB;
		temp[ai] = dev_b[ai];
		temp[bi] = dev_b[bi];

		printf("ai[%d]: %f\t", ai, dev_b[ai]);
		printf("bi[%d]: %f\t", bi, dev_b[bi]);
		printf("Total Offset: %d, %d\n", ai, bi);
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree  
	{
		__syncthreads();
		if (thid < d)
		{//thid starts from 0
			int ai2 = offset*(2 * thid + 1) - 1;
			int bi2 = offset*(2 * thid + 2) - 1;
			//ai += CONFLICT_FREE_OFFSET(ai);
			//bi += CONFLICT_FREE_OFFSET(bi);
			//printf("thid %d, temp[%d] += temp[%d]\n",thid, bi2, ai2);
			temp[bi2] += temp[ai2];
		}
		offset *= 2;

	}
		if (thid == 0) { temp[n - 1] = 0; }
		for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
		{
			offset >>= 1;
			__syncthreads();
			if (thid < d)
			{
				int ai3 = offset*(2 * thid + 1) - 1;
				int bi3 = offset*(2 * thid + 2) - 1;
				//ai += CONFLICT_FREE_OFFSET(ai);
				//bi += CONFLICT_FREE_OFFSET(bi);
				float t = temp[ai3];
				temp[ai3] = temp[bi3];
				temp[bi3] += t;
			}
		}
		__syncthreads();
		dev_c[2*thid] = temp[2 * thid];
		dev_c[2 * thid + 1] = temp[2 * thid + 1];

		printf("Index %d--: %f, %f\n",2*thid,  temp[2 * thid], temp[2 * thid + 1]);


}

__global__ void index_repeats(float *dev_b, float *dev_c, float *dev_B, float n) {


}
__global__ void remove_repeats(float *dev_a, float *dev_B, float *dev_C, float n) {


}

main() {

	float *a, *b, *c; //host ABC
	float *dev_a, *dev_b, *dev_c, *dev_B, *dev_C;
	int size = N_SIZE * sizeof(float);
	int i;
	float toAdd;

	//cudamalloc a, b, and c on device memory

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	hipMalloc((void**)&dev_B, size);
	hipMalloc((void**)&dev_C, size);

	a = (float *)malloc(size);
	b = (float *)malloc(size);
	c = (float *)malloc(size);

	//Make random array
	for (i = 0; i < N_SIZE; i++) {
		a[i] = (float)(rand() % 10);
	}

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	//Do find_repeats, store in dev_b
	find_repeats <<< N_SIZE / 2 / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_a, dev_b, N_SIZE);

	hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);

	//Do exclusive scan on dev_b, store in dev_c
	exclusive_scan <<< N_SIZE / 2 / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_b, dev_c, N_SIZE);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);


	for (i = (N_SIZE / THREADS_PER_BLOCK) - 1; i > 0; i--)
	{
		// Starting from the last block, add the last number from the previous block
		// End one block before the first block
		// Note that each block in the previous kernel calls handled 2*THREADS_PER_BLOCK elements
		// Each of these blocks handles only THREADS_PER_BLOCK elements. 
		toAdd = c[THREADS_PER_BLOCK*i - 1];
		addToAll <<< (N_SIZE / THREADS_PER_BLOCK) - i, THREADS_PER_BLOCK >>> (&dev_c[THREADS_PER_BLOCK * i], toAdd);
	}

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);


	//Create B and C with some cuda operations on dev_c

	//index_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_b, dev_c, dev_B, N_SIZE);

	//remove_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_B, dev_C, N_SIZE);


	printf("Last element of find_repeats: %d\n", 0); //replace

	FILE *ff = fopen("A.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(ff, "%f ", a[i]);
	}

	FILE *f = fopen("B.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(f, "%f ", b[i]);
	}

	FILE *fff = fopen("C.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(fff, "%f ", c[i]);
	}


	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(a); free(b); free(c);
	return 0;

}



