#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "matrix_gen.h"

//Cuda implementation of prefix finder
//Structure from NVIDIA tutorial slides
#define N_SIZE 32
//N_SIZE must be a power of 2
#define THREADS_PER_BLOCK 8
//THREADS_PER_BLOCK must also be a power of 2
//Following 3 defines are for if conflict-free indexing is later implemented
//Not currently in use
#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

__global__ void find_repeats(int *dev_a, int *dev_b, int n) {
	// Places a 1 in dev_b[i] whenever dev_a[i] == dev_a[i+1]
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) { // Don't check the last element, you'll go out of bounds.
		if (dev_a[index] == dev_a[index + 1]) {
			dev_b[index] = 1;
		}
		else {
			dev_b[index] = 0;
		}
	}
}
__global__ void exclusive_scan(int *dev_b, int *dev_c, int n) {
	__shared__ int temp[THREADS_PER_BLOCK];  // allocated on invocation  
	//int nn = blockDim.x;
	int thid = threadIdx.x % blockDim.x;
	// We perform exclusive scan independently on every block in dev_b in place, storing the result in dev_c
	// To seperate each block, we mod the thread ID by blockDim, 
	// and pretend that we're working with only one array that fits within a single block
	int offset_tid = threadIdx.x + blockIdx.x * blockDim.x;
	// To write the data back to dev_c, we do need the offset in the big picture though.
	int offset = 1; 
	int ai = 2*offset_tid;
	int bi = 2* offset_tid + 1;// +(n / 2);
	//We need to use offset_tid to read the data from dev_b as well
	if (bi < n) {
		//We only want to work with data as long as bi+1 is within the range of the input data

		//int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
		//int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
		//ai += bankOffsetA;
		//bi += bankOffsetB;
		temp[ai] = dev_b[ai];
		temp[bi] = dev_b[bi];
		printf("temp[%d]: %d\t", ai, dev_b[ai]);
		printf("temp[%d]: %d\t", bi, dev_b[bi]);
		printf("Total Offset: %d, %d\n", ai, bi);

		for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree  
		{ // We loop once for every row of the binary tree.
			__syncthreads();
			if (thid < d) // This is the modded thread index. 
			{//thid starts from 0
				int ai2 = offset*(2 * thid + 1) - 1;
				int bi2 = offset*(2 * thid + 2) - 1;
				//ai += CONFLICT_FREE_OFFSET(ai);
				//bi += CONFLICT_FREE_OFFSET(bi);
				printf("thid %d, temp[%d] += temp[%d]; %d += %d \n", thid, bi2, ai2, temp[bi2], temp[ai2]);
				temp[bi2] += temp[ai2];
				//On every row, we will add two nodes together, and store it in place on the same array
				//See GPU Gems on the NVIDIA website for visuals.
			}
			offset *= 2;
			//The referenced nodes have a greater offset for each subsequent level

		}
		//if (offset_tid== N_SIZE/2-1) { temp[n - 1] = 0; } // On the very last block, we zero out the last element, only once
		if (thid == 0) { temp[n - 1] = 0; } // Above is a change that I think might be needed, along with a few more changes from n to nn
		//This would be to handle blocks being distinct prefix sums... But there would be a boundary problem, 
		// especially if each block has a zeroed last element
		for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
		{
			offset >>= 1;
			__syncthreads();
			if (thid < d)
			{
				int ai3 = offset*(2 * thid + 1) - 1;
				int bi3 = offset*(2 * thid + 2) - 1;
				//ai += CONFLICT_FREE_OFFSET(ai);
				//bi += CONFLICT_FREE_OFFSET(bi);
				int t = temp[ai3];
				temp[ai3] = temp[bi3];
				temp[bi3] += t;
			}
		}
		__syncthreads();
		dev_c[2 * offset_tid] = temp[2 * thid];
		dev_c[2 * offset_tid + 1] = temp[2 * thid + 1];
		//Push the data back to global memory
		//We use shared memory because it's very much faster.
		printf("Index %d--: %d, %d\n", 2 * thid, temp[2 * thid], temp[2 * thid + 1]);
	}
}

__global__ void addToAll(int *dev_c, int n, int *dev_toAdd) {
	//Add offset to all elements, so it becomes as though we had done a prefix sum over the entire array
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < n) {
		dev_c[index] += dev_toAdd[blockIdx.x];
	}
}

__global__ void index_repeats(int *dev_c, int *dev_B, int n) {
	// Find the index of the repeating elements.
	// The index is found by the prefix sum, and the largest element is discovered in addToAll
	// We create an array to hand to index_repeats of size according to largest element
	// When we find an inequality in the prefix sum, the value is the index of dev_B, and the index is the value of dev_B
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n-1)
	{
		if (dev_c[index] != dev_c[index + 1])
		{
			dev_B[dev_c[index]] = index;
		}
	}
}
__global__ void remove_repeats(int *dev_a, int *dev_B, int *dev_C, int n) {
	//Replace the indexes in dev_B with values by referencing dev_a
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) {
		dev_C[index] = dev_a[dev_B[index]];
	}
}

main() {

	int *a, *b, *c, *B, *C; //host ABC
	int *dev_a, *dev_b, *dev_c, *dev_B, *dev_C, *dev_toAdd;
	int size = N_SIZE * sizeof(int);
	printf("size is %d\n", size);
	int i;
	int *toAdd;
	int numRepeats;

	//cudamalloc a, b, and c on device memory

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	//Make random array
	for (i = 0; i < N_SIZE; i++) {
		a[i] = (int)(rand() % 10);
	}

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	//Do find_repeats, store in dev_b
	find_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_a, dev_b, N_SIZE);

	hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);

	//Do exclusive scan on dev_b, store in dev_c
	exclusive_scan <<< (N_SIZE / 2) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_b, dev_c, N_SIZE);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	//We now know the largest element in dev_c.
	// That means we know how many repeats there are.
	int run = 0;
	int temp = 0;
	int size2 = sizeof(int) * (N_SIZE / THREADS_PER_BLOCK);
	toAdd = (int*)malloc(size2);
	hipMalloc((void**)&dev_toAdd, size2);
	toAdd[0] = 0;
	//Here, we find the largest element of each of the parallel prefix sums, and sum them
	// The calculated offset for each block is stored in a toAdd array
	for (i = 1; i < (N_SIZE / THREADS_PER_BLOCK); i++)
	{
		temp = THREADS_PER_BLOCK*i - 1;
		printf("temp is: %d\n", temp);
		if (temp < N_SIZE) {
			run += c[temp];
			toAdd[i] = run;
			printf("toAdd[%d] = run is: %d\n",i, run);
		}
	}
	hipMemcpy(dev_toAdd, toAdd, size2, hipMemcpyHostToDevice);
	addToAll <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_c, N_SIZE, dev_toAdd);
	// dev_c array should contain proper data now.
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	
	numRepeats = c[N_SIZE - 1];
	printf("Number of Repeats: %d\n", numRepeats); 
	B = (int*)malloc(sizeof(int)*numRepeats);
	C = (int*)malloc(sizeof(int)*numRepeats);
	hipMalloc((void**)&dev_B, sizeof(int)*numRepeats);
	hipMalloc((void**)&dev_C, sizeof(int)*numRepeats);

	//Create B and C with some cuda operations on dev_c

	index_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_c, dev_B, N_SIZE);

	hipMemcpy(B, dev_B, sizeof(int)*numRepeats, hipMemcpyDeviceToHost);
	int thisNumBlocks = 1;
	if( numRepeats > THREADS_PER_BLOCK)
		thisNumBlocks = numRepeats / THREADS_PER_BLOCK;

	remove_repeats <<< thisNumBlocks, THREADS_PER_BLOCK >>>(dev_a, dev_B, dev_C, numRepeats);

	hipMemcpy(C, dev_C, sizeof(int)*numRepeats, hipMemcpyDeviceToHost);


	FILE *ff = fopen("A.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(ff, "%d ", a[i]);
	}

	FILE *f = fopen("B.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(f, "%d ", b[i]);
	}

	FILE *fff = fopen("C.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(fff, "%d ", c[i]);
	}

	FILE *ffff = fopen("BB.arr", "wb");
	for (i = 0; i < numRepeats; i++) {
		fprintf(ffff, "%d ", B[i]);
	}

	FILE *fffff = fopen("CC.arr", "wb");
	for (i = 0; i < numRepeats; i++) {
		fprintf(fffff, "%d ", C[i]);
	}


	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_B);
	hipFree(dev_C);
	free(a); free(b); free(c); free(C); free(B); free(toAdd);
	return 0;

}



