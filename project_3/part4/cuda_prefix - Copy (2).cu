#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "matrix_gen.h"

//Cuda implementation of prefix finder
//Structure from NVIDIA tutorial slides
#define N_SIZE 30
#define THREADS_PER_BLOCK 8
#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

__global__ void find_repeats(int *dev_a, int *dev_b, int n) {

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) {
		if (dev_a[index] == dev_a[index + 1]) {
			dev_b[index] = 1;
		}
		else {
			dev_b[index] = 0;
		}
	}
}
__global__ void addToAll(int *dev_c, int n, int *dev_toAdd) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < n) {
		dev_c[index] += dev_toAdd[blockIdx.x];
	}
}

__global__ void exclusive_scan(int *dev_b, int *dev_c, int n) {
	__shared__ int temp[THREADS_PER_BLOCK];  // allocated on invocation  
	int thid = threadIdx.x % blockDim.x;
	int offset_tid = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = 1; 
	int ai = 2*offset_tid;
	int bi = 2* offset_tid + 1;// +(n / 2);
	if (bi < n) {
		//int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
		//int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
		//ai += bankOffsetA;
		//bi += bankOffsetB;
		temp[ai] = dev_b[ai];
		temp[bi] = dev_b[bi];
		printf("temp[%d]: %d\t", ai, dev_b[ai]);
		printf("temp[%d]: %d\t", bi, dev_b[bi]);
		printf("Total Offset: %d, %d\n", ai, bi);

		for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree  
		{
			__syncthreads();
			if (thid < d)
			{//thid starts from 0
				int ai2 = offset*(2 * thid + 1) - 1;
				int bi2 = offset*(2 * thid + 2) - 1;
				//ai += CONFLICT_FREE_OFFSET(ai);
				//bi += CONFLICT_FREE_OFFSET(bi);
				printf("thid %d, temp[%d] += temp[%d]; %d += %d \n", thid, bi2, ai2, temp[bi2], temp[ai2]);
				temp[bi2] += temp[ai2];
			}
			offset *= 2;

		}
		if (thid == 0) { temp[n - 1] = 0; }
		for (int d = 1; d < n; d *= 2) // traverse down tree & build scan  
		{
			offset >>= 1;
			__syncthreads();
			if (thid < d)
			{
				int ai3 = offset*(2 * thid + 1) - 1;
				int bi3 = offset*(2 * thid + 2) - 1;
				//ai += CONFLICT_FREE_OFFSET(ai);
				//bi += CONFLICT_FREE_OFFSET(bi);
				int t = temp[ai3];
				temp[ai3] = temp[bi3];
				temp[bi3] += t;
			}
		}
		__syncthreads();
		dev_c[2 * thid] = temp[2 * thid];
		dev_c[2 * thid + 1] = temp[2 * thid + 1];
		printf("Index %d--: %d, %d\n", 2 * thid, temp[2 * thid], temp[2 * thid + 1]);
	}
}

__global__ void index_repeats(int *dev_c, int *dev_B, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n-1)
	{
		if (dev_c[index] != dev_c[index + 1])
		{
			dev_B[dev_c[index]] = index;
		}
	}
}
__global__ void remove_repeats(int *dev_a, int *dev_B, int *dev_C, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) {
		dev_C[index] = dev_a[dev_B[index]];
	}
}

main() {

	int *a, *b, *c, *B, *C; //host ABC
	int *dev_a, *dev_b, *dev_c, *dev_B, *dev_C, *dev_toAdd;
	int size = N_SIZE * sizeof(int);
	printf("size is %d\n", size);
	int i;
	int *toAdd;
	int numRepeats;

	//cudamalloc a, b, and c on device memory

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	//Make random array
	for (i = 0; i < N_SIZE; i++) {
		a[i] = (int)(rand() % 10);
	}

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	//Do find_repeats, store in dev_b
	find_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_a, dev_b, N_SIZE);

	hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);

	//Do exclusive scan on dev_b, store in dev_c
	exclusive_scan <<< (N_SIZE / 2) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_b, dev_c, N_SIZE);

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	int run = 0;
	int temp = 0;
	int size2 = sizeof(int) * (N_SIZE / THREADS_PER_BLOCK);
	toAdd = (int*)malloc(size2);
	hipMalloc((void**)&dev_toAdd, size2);
	toAdd[0] = 0;
	for (i = 1; i < (N_SIZE / THREADS_PER_BLOCK); i++)
	{
		temp = THREADS_PER_BLOCK*i - 1;
		printf("temp is: %d\n", temp);
		if (temp < N_SIZE) {
			run += c[temp];
			toAdd[i] = run;
			printf("toAdd[%d] = run is: %d\n",i, run);
		}
	}
	hipMemcpy(dev_toAdd, toAdd, size2, hipMemcpyHostToDevice);
	printf("Start addToAll");
	addToAll <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_c, N_SIZE, dev_toAdd);
	printf("End addToAll");
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	
	numRepeats = c[N_SIZE - 1];
	printf("Number of Repeats: %d\n", numRepeats); 
	B = (int*)malloc(sizeof(int)*numRepeats);
	C = (int*)malloc(sizeof(int)*numRepeats);
	hipMalloc((void**)&dev_B, sizeof(int)*numRepeats);
	hipMalloc((void**)&dev_C, sizeof(int)*numRepeats);

	//Create B and C with some cuda operations on dev_c

	index_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_c, dev_B, N_SIZE);

	hipMemcpy(B, dev_B, sizeof(int)*numRepeats, hipMemcpyDeviceToHost);
	int thisNumBlocks = 1;
	if( numRepeats > THREADS_PER_BLOCK)
		thisNumBlocks = numRepeats / THREADS_PER_BLOCK;
	remove_repeats <<< thisNumBlocks, THREADS_PER_BLOCK >>>(dev_a, dev_B, dev_C, numRepeats);

	hipMemcpy(C, dev_C, sizeof(int)*numRepeats, hipMemcpyDeviceToHost);


	FILE *ff = fopen("A.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(ff, "%d ", a[i]);
	}

	FILE *f = fopen("B.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(f, "%d ", b[i]);
	}

	FILE *fff = fopen("C.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(fff, "%d ", c[i]);
	}

	FILE *ffff = fopen("BB.arr", "wb");
	for (i = 0; i < numRepeats; i++) {
		fprintf(ffff, "%d ", B[i]);
	}

	FILE *fffff = fopen("CC.arr", "wb");
	for (i = 0; i < numRepeats; i++) {
		fprintf(fffff, "%d ", C[i]);
	}


	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_B);
	hipFree(dev_C);
	free(a); free(b); free(c); free(C); free(B); free(toAdd);
	return 0;

}



