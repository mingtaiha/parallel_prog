#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//Cuda implementation of prefix finder
//Structure from NVIDIA tutorial slides
#define N_SIZE 1048576
//N_SIZE must be a power of 2
#define THREADS_PER_BLOCK 512
//THREADS_PER_BLOCK must also be a power of 2
//Following 3 defines are for if conflict-free indexing is later implemented
//Not currently in use
#define NUM_BANKS 16  
#define LOG_NUM_BANKS 4  
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

__global__ void find_repeats(int *dev_a, int *dev_b, int n) {
	// Places a 1 in dev_b[i] whenever dev_a[i] == dev_a[i+1]
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n - 1) { // Don't check the last element, you'll go out of bounds.
		if (dev_a[index] == dev_a[index + 1]) {
			dev_b[index] = 1;
		}
		else {
			dev_b[index] = 0;
		}
	}
}
__global__ void exclusive_scan(int *dev_b, int *dev_c, int *dev_toAdd, int n) {
	__shared__ int temp[THREADS_PER_BLOCK*2];  // allocated on invocation  
	int nn = blockDim.x*2;
	int thid = threadIdx.x % blockDim.x;
	// We perform exclusive scan independently on every block in dev_b in place, storing the result in dev_c
	// To seperate each block, we mod the thread ID by blockDim, 
	// and pretend that we're working with only one array that fits within a single block
	int offset_tid = threadIdx.x + blockIdx.x * blockDim.x;
	// To write the data back to dev_c, we do need the offset in the big picture though.
	int offset = 1;
	int ai = 2 * offset_tid;
	int bi = 2 * offset_tid + 1;// +(n / 2);
	int ai0 = 2 * thid;
	int bi0 = 2 * thid + 1;
	//We need to use offset_tid to read the data from dev_b as well

		//int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
		//int bankOffsetB = CONFLICT_FREE_OFFSET(bi); 
		//ai += bankOffsetA;
		//bi += bankOffsetB;
	temp[ai0] = dev_b[ai];
	temp[bi0] = dev_b[bi];
	//printf("temp[%d]: %d\ttemp[%d]: %d\n", ai0, dev_b[ai], bi0, dev_b[bi]);

	for (int d = nn >> 1; d > 0; d >>= 1) // build sum in place up the tree  
	{ // We loop once for every row of the binary tree.
		__syncthreads();
		if (thid < d) // This is the modded thread index. 
		{//thid starts from 0
			int ai2 = offset*(2 * thid + 1) - 1;
			int bi2 = offset*(2 * thid + 2) - 1;
			//ai += CONFLICT_FREE_OFFSET(ai);
			//bi += CONFLICT_FREE_OFFSET(bi);
			//printf("thid %d, temp[%d] += temp[%d]\t%d += %d \n", offset_tid, bi2, ai2, temp[bi2], temp[ai2]);
			temp[bi2] += temp[ai2];
			//On every row, we will add two nodes together, and store it in place on the same array
			//See GPU Gems on the NVIDIA website for visuals.
		}
		offset *= 2;
		//The referenced nodes have a greater offset for each subsequent level

	}
	if (thid == 0) {
		dev_toAdd[blockIdx.x] = temp[nn - 1];
		//printf("dev_toAdd[%d], %d at index %d\n", blockIdx.x, temp[nn - 1], nn - 1);
		temp[nn - 1] = 0;
	}
	//Store the sum before zeroing
	for (int d = 1; d < nn; d *= 2) // traverse down tree & build scan  
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai3 = offset*(2 * thid + 1) - 1;
			int bi3 = offset*(2 * thid + 2) - 1;
			//ai += CONFLICT_FREE_OFFSET(ai);
			//bi += CONFLICT_FREE_OFFSET(bi);
			int t = temp[ai3];
			temp[ai3] = temp[bi3];
			temp[bi3] += t;
		}
	}
	__syncthreads();
	dev_c[ai] = temp[ai0];
	dev_c[bi] = temp[bi0];
	//Push the data back to global memory
	//We use shared memory because it's very much faster.
	//printf("Index %d--: %d, %d\n", ai0, temp[ai0], temp[bi0]);
}
__global__ void addToAll(int *dev_c, int n, int *dev_toAdd) {
	//Add offset to all elements, so it becomes as though we had done a prefix sum over the entire array
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	if (index < n) {
		dev_c[index] += dev_toAdd[(int)(blockIdx.x / 2)];
	}
}

__global__ void index_repeats(int *dev_c, int *dev_B, int n) {
	// Find the index of the repeating elements.
	// The index is found by the prefix sum, and the largest element is discovered in addToAll
	// We create an array to hand to index_repeats of size according to largest element
	// When we find an inequality in the prefix sum, the value is the index of dev_B, and the index is the value of dev_B
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n-1)
	{
		if (dev_c[index] != dev_c[index + 1])
		{
			dev_B[dev_c[index]] = index;
		}
	}
}
__global__ void remove_repeats(int *dev_a, int *dev_c, int *dev_C, int n) {
	//Replace the indexes in dev_B with values by referencing dev_a
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n ) {
		dev_C[index] = dev_a[index - dev_c[index]];
	}
}

main() {

	int *a, *b, *c, *B, *C; //host ABC
	int *dev_a, *dev_b, *dev_c, *dev_B, *dev_C, *dev_toAdd, *dev_toAdd0, *dev_throwaway;
	int size = N_SIZE * sizeof(int);
	//printf("size is %d\n", size);
	int i;
	int *toAdd;
	int numRepeats;
	int size2 = sizeof(int) * (N_SIZE / 2 / THREADS_PER_BLOCK);


	//cudamalloc a, b, and c on device memory

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	hipMalloc((void**)&dev_toAdd, size2);
	hipMalloc((void**)&dev_toAdd0, size2);
	hipMalloc((void**)&dev_throwaway, sizeof(int));

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);
	toAdd = (int *)malloc(size2);

	//Make random array
	for (i = 0; i < N_SIZE; i++) {
		a[i] = (int)(rand() % 10);
	}

	clock_t begin = clock();
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);

	//Do find_repeats, store in dev_b
	find_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_a, dev_b, N_SIZE);

	hipMemcpy(b, dev_b, size, hipMemcpyDeviceToHost);

	//Do exclusive scan on dev_b, store in dev_c
	exclusive_scan <<< (N_SIZE / 2) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_b, dev_c, dev_toAdd0, N_SIZE);

	//hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	//hipMemcpy(toAdd, dev_toAdd0, size2, hipMemcpyDeviceToHost);

	//We now know the largest element in dev_c.
	// That means we know how many repeats there are.

	//We stored the largest element of each of the parallel prefix sums, and will now sum them
	// The calculated offset for each block is stored in a toAdd array

	exclusive_scan <<< 1, N_SIZE / 4 / THREADS_PER_BLOCK >>> (dev_toAdd0, dev_toAdd, dev_throwaway, N_SIZE / 2 / THREADS_PER_BLOCK);
	/*
	printf("toAdd0: %d\n", toAdd[0]);
	for (i = 1; i < N_SIZE/ 2 / THREADS_PER_BLOCK; i++)
	{//Iterate over each block
		toAdd[i] += toAdd[i - 1];
		printf("toAdd: %d\n",toAdd[i]);
	}
	toAdd[0] = 0;
	hipMemcpy(dev_toAdd, toAdd, size2, hipMemcpyHostToDevice);*/
	addToAll <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_c, N_SIZE, dev_toAdd);
	// dev_c array should contain proper data now.
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	
	numRepeats = c[N_SIZE - 1];
	printf("Number of Repeats: %d\n", numRepeats); 
	B = (int*)malloc(sizeof(int)*numRepeats);
	hipMalloc((void**)&dev_B, sizeof(int)*numRepeats);

	//Create B and C with some cuda operations on dev_c

	index_repeats <<< N_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_c, dev_B, N_SIZE);

	hipMemcpy(B, dev_B, sizeof(int)*numRepeats, hipMemcpyDeviceToHost);
	int thisNumBlocks = 1;
	if( numRepeats > THREADS_PER_BLOCK)
		thisNumBlocks = numRepeats / THREADS_PER_BLOCK;

	int nonRepeats = N_SIZE - numRepeats;
	int size3 = nonRepeats * sizeof(int);
	C = (int*)malloc(size3);
	hipMalloc((void**)&dev_C, size3);



	remove_repeats <<< thisNumBlocks, THREADS_PER_BLOCK >>>(dev_a, dev_c, dev_C, numRepeats);

	hipMemcpy(C, dev_C, sizeof(int)*numRepeats, hipMemcpyDeviceToHost);


	clock_t end = clock();

	float timeTaken = (float)((end - begin) / (float)CLOCKS_PER_SEC);

	printf("time taken: %f\n", timeTaken);
	FILE *ff = fopen("A.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(ff, "%d ", a[i]);
	}

	FILE *f = fopen("B.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(f, "%d ", b[i]);
	}

	FILE *fff = fopen("C.arr", "wb");
	for (i = 0; i < N_SIZE; i++) {
		fprintf(fff, "%d ", c[i]);
	}

	FILE *ffff = fopen("BB.arr", "wb");
	for (i = 0; i < numRepeats; i++) {
		fprintf(ffff, "%d ", B[i]);
	}

	FILE *fffff = fopen("CC.arr", "wb");
	for (i = 0; i < numRepeats; i++) {
		fprintf(fffff, "%d ", C[i]);
	}


	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_B);
	hipFree(dev_C);
	free(a); free(b); free(c); free(C); free(B); free(toAdd);
	return 0;

}



