#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "problem1.h"


__global__ void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

void hello_gpu(char *a, int *b, const int N, const int blocksize) {

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc( (void**)&ad, csize );
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);

	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );
	hipFree( bd );
	printf("%s\n", a);
	sleep(1);
}


__global__ void kernel_max(double * array, double * max, long int num_elem)
{

	/*
	extern __shared__ double sdata[];
	unsigned long int tid = threadIdx.x;
	unsigned long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	
	sdata[tid] = array[idx];
	//unsigned int i = (unsigned int)ceil((float)blockDim.x/2);
		__syncthreads();

	unsigned long int i;
	for ( i = 1; i < blockDim.x; i*=2)
	{
		if ((tid % (2*i) == 0) && (idx + i < num_elem)) 
		{
			sdata[tid] = (sdata[tid] > sdata[tid + i]) ? sdata[tid] : sdata[tid + i];
		}
		__syncthreads();
	}
	__syncthreads();

	if ( tid == 0 )
	{
		max[blockIdx.x] = sdata[tid];
	}
	*/

	
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	max[idx] = array[idx];

	long int i;
	for (i = 1; i < num_elem ; i*=2)
	{
		if ((idx % (2*i) == 0) && (idx + i < num_elem))
		{
			max[idx] = (max[idx] > max[idx + i]) ? max[idx] : max[idx + i];
		}
		__syncthreads();
	}
	__syncthreads();
	
}

void max_gpu(double *array, long int size, int tpb)
{

	long int nblocks = (long int)ceil((double)size / (double)tpb);
	printf("N blocks: %ld\n", nblocks);

	double *dev_array, *dev_array2;
	double *dev_max_array;
	double *dev_final_max_array;
	long int arr_size = size * sizeof(double);

	double *max_array = (double *) malloc(nblocks * sizeof(double));
	//double *output_array;
	//max_array[0] = 11.11;
	printf("%f\n", max_array[0]);


	printf("hipMalloc'ing\n");
	hipMalloc( (void**)&dev_array, arr_size);
	hipMalloc( (void**)&dev_array2, arr_size);
	hipMalloc( (void**)&dev_max_array, nblocks * sizeof(double));
	hipMalloc( (void**)&dev_final_max_array, nblocks * sizeof(double));

	//printf("%f\n", array[0]);
	printf("hipMemcpy to device\n");
	hipMemcpy(dev_array, array, arr_size, hipMemcpyHostToDevice);
	//hipMemcpy(arr2, dev_array, arr_size, hipMemcpyDeviceToHost);
	//printf("%f\n", arr2[0]);
	hipMemcpy(dev_max_array, max_array, nblocks * sizeof(double), hipMemcpyHostToDevice);	

	//printf("Calling kernel_max\n");
	
	int count = 0;
	//printf("Calling kernel_max\n");
	while (nblocks/tpb >= 1) {
		printf("Calling kernel_max\n");
		if (count % 2 == 0) {
			kernel_max<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array, dev_array2, size);
		}
		else
		{
			kernel_max<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array2, dev_array, size);
		}
	//kernel_max<<<dimGrid, tpb, tpb * sizeof(double)>>>(dev_array, dev_max_array, size);
		hipDeviceSynchronize();
		count++;
		size = nblocks;
		nblocks = (long int)ceil((double)size / double(tpb));
	}
	//hipError_t code = hipMemcpy(max_array, dev_max_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	//hipError_t code = hipMemcpy(array, dev_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	//int i = 0;
	//for(; i < nblocks; i++) { printf("%f\n", max_array[i]); }
	
	//printf("Calling reduced kernel_max\n");
	//printf("Calling reduced kernel_max\n");
	//kernel_max<<<1, dimGrid, tpb * sizeof(double)>>>(dev_max_array, dev_final_max_array, nblocks);
	//hipDeviceSynchronize();
	
	hipError_t code;
	if (count % 2 == 0) {	
		code = hipMemcpy(max_array, dev_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}
	else
	{
		code = hipMemcpy(max_array, dev_array2, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}
	//i = 0;	
	//for(; i < nblocks; i++) { printf("%f\n", max_array[i]); }
	//printf("hipMemcpy to host\n");
	
	//hipError_t code = hipMemcpy(max_array, dev_max_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	printf("Code: %d\n", code);
	printf( "Max: %f\n", max_array[0] );

	//i = 0;
	//for(; i < nblocks; i++) { printf("%f\n", max_array[i]); }
	free( max_array );
	hipFree( dev_array );
	hipFree( dev_max_array );

}
