#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "problem1.h"



__global__ void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

void hello_gpu(char *a, int *b, const int N, const int blocksize) {

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc( (void**)&ad, csize );
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);

	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );
	hipFree( bd );
	printf("%s\n", a);
	sleep(1);
}


__global__ void kernel_max(double * array, double * max, int num_elem)
{

	extern __shared__ double sdata[];
	int tid = threadIdx.x;
	int idx = blockIdx.x - 1* blockDim.x + threadIdx.x;

	//sdata[tid] = 5.0;
	sdata[tid] = array[idx];
	int i = (unsigned int)ceil((float)blockDim.x/2);

	__syncthreads();

	/*
	int i;
	for ( i = 1; i < blockDim.x; i*=2)
	{
		if ((tid % (2*i) == 0) && (idx + i < num_elem)) {
			sdata[tid] = (sdata[tid] > sdata[tid + i]) ? sdata[tid] : sdata[tid + i];
		}
	}
	*/

		
	while( i!=0 )
	{
		if ( tid+i < num_elem && idx < i )
		{
			sdata[idx] = (sdata[idx] > sdata[idx+i]) ? sdata[idx] : sdata[idx+i];
		}
		i /= 2;
	}
	
	
	__syncthreads();

	
	if ( tid == 0 )
	{
		max[blockIdx.x] = sdata[0];
	}
	
	
}

void max_gpu(double *array, int size, int tpb)
{

	int nblocks = (int)ceil((double)size / (double)tpb);

	double *dev_array;
	double *dev_max_array;
	int arr_size = size * sizeof(double);

	//double *arr2 = (double *) malloc(size * sizeof(double));
	double *max_array = (double *) malloc(nblocks * sizeof(double));
	//double *output_array;
	//max_array[0] = 11.11;
	printf("%f\n", max_array[0]);


	printf("hipMalloc'ing\n");
	hipMalloc( (void**)&dev_array, arr_size);
	hipMalloc( (void**)&dev_max_array, 1 * sizeof(double));

	printf("%f\n", array[0]);
	printf("hipMemcpy to device\n");
	hipMemcpy(dev_array, array, arr_size, hipMemcpyHostToDevice);
	//hipMemcpy(arr2, dev_array, arr_size, hipMemcpyDeviceToHost);
	//printf("%f\n", arr2[0]);
	hipMemcpy(dev_max_array, max_array, nblocks * sizeof(double), hipMemcpyHostToDevice);	

	printf("Calling kernel_max\n");
	
	dim3 dimGrid(16, 1);
	dim3 dimBlock(32, 1);
	kernel_max<<<dimGrid, dimBlock>>>(dev_array, dev_max_array, size);
	//hipDeviceSynchronize();
	
	//printf("hipMemcpy to host\n");
	
	hipError_t code = hipMemcpy(max_array, dev_max_array, 1 * sizeof(double), hipMemcpyDeviceToHost);
	printf("%d\n", code);
	printf( "Max: %f\n", max_array[0] );

	free( max_array );
	hipFree( dev_array );
	hipFree( dev_max_array );

}
