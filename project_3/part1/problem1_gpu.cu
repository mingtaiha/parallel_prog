#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "problem1.h"


__global__ void hello(char *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

void hello_gpu(char *a, int *b, const int N, const int blocksize) {

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc( (void**)&ad, csize );
	hipMalloc( (void**)&bd, isize );
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);

	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
	hipFree( ad );
	hipFree( bd );
	printf("%s\n", a);
	sleep(1);
}


__global__ void kernel_max(double * array, double * max, long int num_elem)
{

	
	extern __shared__ double sdata[];
	unsigned long int tid = threadIdx.x;
	unsigned long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	
	sdata[tid] = array[idx];
	//unsigned int i = (unsigned int)ceil((float)blockDim.x/2);
	__syncthreads();

	unsigned long int i;
	for ( i = 1; i < blockDim.x; i*=2)
	{
		if ((tid % (2*i) == 0) && (idx + i < num_elem)) 
		{
			sdata[tid] = (sdata[tid] > sdata[tid + i]) ? sdata[tid] : sdata[tid + i];
		}
		__syncthreads();
	}
	__syncthreads();

	if ( tid == 0 )
	{
		max[blockIdx.x] = sdata[tid];
	}
	

/*
	long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	max[idx] = array[idx];

	__syncthreads();

	long int i;
	for (i = 1; i < num_elem ; i*=2)
	{
		if ((idx % (2*i) == 0) && (idx + i < num_elem))
		{
			max[idx] = (max[idx] > max[idx + i]) ? max[idx] : max[idx + i];
		}
		__syncthreads();
	}
	__syncthreads();
*/	
}

void max_gpu(double *array, long int size, int tpb)
{

	long int nblocks = (long int)ceil((double)size / (double)tpb);
	//printf("N blocks: %ld\n", nblocks);

	double *dev_array, *dev_array2;
	double *dev_max_array;
	long int arr_size = size * sizeof(double);

	double *max_array = (double *) malloc(size * sizeof(double));
	//printf("%f\n", max_array[0]);


	//printf("hipMalloc'ing\n");
	hipMalloc( (void**)&dev_array, arr_size);
	hipMalloc( (void**)&dev_array2, arr_size);
	hipMalloc( (void**)&dev_max_array, nblocks * sizeof(double));

	//printf("hipMemcpy to device\n");
	hipMemcpy(dev_array, array, arr_size, hipMemcpyHostToDevice);

	clock_t start, end;
	start = clock();

	int count = 0;
	while (nblocks >= 1) {
		//printf("Calling kernel_max\n");
		//printf("nblocks: %ld\n", nblocks);
		if (count % 2 == 0) {
			kernel_max<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array, dev_array2, size);
		}
		else
		{
			kernel_max<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array2, dev_array, size);
		}
		hipDeviceSynchronize();
		if (nblocks == 1) { break; }
		count++;
		size = nblocks;
		nblocks = (long int)ceil((double)size / (double)tpb);
	}
	end = clock();
	printf("Time for Max: %f\n", (((double)end - (double)start)) / CLOCKS_PER_SEC);

	if (count % 2 == 0) {	
		hipMemcpy(max_array, dev_array2, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}
	else
	{
		hipMemcpy(max_array, dev_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}

	printf( "Max: %f\n", max_array[0] );

	free( max_array );
	hipFree( dev_array );
	hipFree( dev_array2 );
	hipFree( dev_max_array );
}



__global__ void kernel_min(double * array, double * max, long int num_elem)
{

	
	extern __shared__ double sdata[];
	unsigned long int tid = threadIdx.x;
	unsigned long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	
	sdata[tid] = array[idx];
	__syncthreads();

	unsigned long int i;
	for ( i = 1; i < blockDim.x; i*=2)
	{
		if ((tid % (2*i) == 0) && (idx + i < num_elem)) 
		{
			sdata[tid] = (sdata[tid] < sdata[tid + i]) ? sdata[tid] : sdata[tid + i];
		}
		__syncthreads();
	}
	__syncthreads();

	if ( tid == 0 )
	{
		max[blockIdx.x] = sdata[tid];
	}
}


void min_gpu(double *array, long int size, int tpb)
{

	long int nblocks = (long int)ceil((double)size / (double)tpb);
	//printf("N blocks: %ld\n", nblocks);

	double *dev_array, *dev_array2;
	double *dev_min_array;
	long int arr_size = size * sizeof(double);

	double *min_array = (double *) malloc(size * sizeof(double));

	//printf("hipMalloc'ing\n");
	hipMalloc( (void**)&dev_array, arr_size);
	hipMalloc( (void**)&dev_array2, arr_size);
	hipMalloc( (void**)&dev_min_array, nblocks * sizeof(double));

	//printf("hipMemcpy to device\n");
	hipMemcpy(dev_array, array, arr_size, hipMemcpyHostToDevice);

	clock_t start, end;
	start = clock();

	int count = 0;
	while (nblocks >= 1) {
		//printf("Calling kernel_min\n");
		//printf("nblocks: %ld\n", nblocks);
		if (count % 2 == 0) {
			kernel_min<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array, dev_array2, size);
		}
		else
		{
			kernel_min<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array2, dev_array, size);
		}
		hipDeviceSynchronize();
		if (nblocks == 1) { break; }
		count++;
		size = nblocks;
		nblocks = (long int)ceil((double)size / (double)tpb);
	}
	end = clock();
	printf("Time for Min: %f\n", (((double)end - (double)start)) / CLOCKS_PER_SEC);

	if (count % 2 == 0) {	
		hipMemcpy(min_array, dev_array2, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}
	else
	{
		hipMemcpy(min_array, dev_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}

	printf( "Min: %f\n", min_array[0] );

	free( min_array );
	hipFree( dev_array );
	hipFree( dev_array2 );
	hipFree( dev_min_array );
}


__global__ void kernel_avg(double * array, double * max, long int num_elem, int final_step)
{

	
	extern __shared__ double sdata[];
	unsigned long int tid = threadIdx.x;
	unsigned long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	
	sdata[tid] = array[idx];
	__syncthreads();

	unsigned long int i;
	for ( i = 1; i < blockDim.x; i*=2)
	{
		if ((tid % (2*i) == 0) && (idx + i < num_elem)) 
		{
			sdata[tid] += sdata[tid + i];
		}
		__syncthreads();
	}
	__syncthreads();

	if ( tid == 0 )
	{
		max[blockIdx.x] = sdata[tid];
		if (final_step == 1) 
		{
			max[blockIdx.x] /= (double)num_elem;
		}
	}
}


void avg_gpu(double *array, long int size, int tpb)
{

	long int nblocks = (long int)ceil((double)size / (double)tpb);
	//printf("N blocks: %ld\n", nblocks);

	double *dev_array, *dev_array2;
	double *dev_avg_array;
	long int arr_size = size * sizeof(double);

	double *avg_array = (double *) malloc(size * sizeof(double));

	//printf("hipMalloc'ing\n");
	hipMalloc( (void**)&dev_array, arr_size);
	hipMalloc( (void**)&dev_array2, arr_size);
	hipMalloc( (void**)&dev_avg_array, nblocks * sizeof(double));

	//printf("hipMemcpy to device\n");
	hipMemcpy(dev_array, array, arr_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_avg_array, avg_array, nblocks * sizeof(double), hipMemcpyHostToDevice);	

	clock_t start, end;
	start = clock();

	int count = 0;
	long int num_elem = size;
	while (nblocks >= 1) {
		//printf("Calling kernel_avg\n");
		//printf("nblocks: %ld\n", nblocks);
		if (count % 2 == 0) {
			kernel_avg<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array, dev_array2, num_elem, nblocks);
		}
		else
		{
			kernel_avg<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array2, dev_array, num_elem, nblocks);
		}
		hipDeviceSynchronize();
		if (nblocks == 1) { break; }
		count++;
		size = nblocks;
		nblocks = (long int)ceil((double)size / (double)tpb);
	}
	end = clock();
	printf("Time for Avg: %f\n", (((double)end - (double)start)) / CLOCKS_PER_SEC);

	if (count % 2 == 0) {	
		hipMemcpy(avg_array, dev_array2, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}
	else
	{
		hipMemcpy(avg_array, dev_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}

	printf( "Avg: %f\n", avg_array[0] );

	free( avg_array );
	hipFree( dev_array );
	hipFree( dev_array2 );
	hipFree( dev_avg_array );
}

__global__ void kernel_std(double * array, double * max, long int num_elem, double avg, int first_step, int final_step)
{

	
	extern __shared__ double sdata[];
	unsigned long int tid = threadIdx.x;
	unsigned long int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (first_step == 0) {
		sdata[tid] = (array[idx] - avg) * (array[idx] - avg);
	}
	else
	{
		sdata[tid] = array[idx];
	}
	__syncthreads();

	unsigned long int i;
	for ( i = 1; i < blockDim.x; i*=2)
	{
		if ((tid % (2*i) == 0) && (idx + i < num_elem)) 
		{
			sdata[tid] += sdata[tid + i];
		}
		__syncthreads();
	}
	__syncthreads();

	if ( tid == 0 )
	{
		max[blockIdx.x] = sdata[tid];
		if (final_step == 1)
		{
			max[blockIdx.x] /= num_elem;
			max[blockIdx.x] = sqrt(max[blockIdx.x]);
		}
	}
}


void std_gpu(double *array, long int size, int tpb)
{

	long int nblocks = (long int)ceil((double)size / (double)tpb);
	//printf("N blocks: %ld\n", nblocks);

	double *dev_array, *dev_array2;
	double *dev_avg_array;
	long int arr_size = size * sizeof(double);

	double *avg_array = (double *) malloc(size * sizeof(double));

	//printf("hipMalloc'ing\n");
	hipMalloc( (void**)&dev_array, arr_size);
	hipMalloc( (void**)&dev_array2, arr_size);
	hipMalloc( (void**)&dev_avg_array, nblocks * sizeof(double));

	//printf("hipMemcpy to device\n");
	hipMemcpy(dev_array, array, arr_size, hipMemcpyHostToDevice);

	clock_t avg_start, avg_end, std_start, std_end;
	avg_start = clock();

	int count = 0;
	long int num_elem = size;
	while (nblocks >= 1) {
		//printf("Calling kernel_avg\n");
		//printf("nblocks: %ld\n", nblocks);
		if (count % 2 == 0) {
			kernel_avg<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array, dev_array2, num_elem, nblocks);
		}
		else
		{
			kernel_avg<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array2, dev_array, num_elem, nblocks);
		}
		hipDeviceSynchronize();
		if (nblocks == 1) { break; }
		count++;
		size = nblocks;
		nblocks = (long int)ceil((double)size / (double)tpb);
	}
	avg_end = clock();


	if (count % 2 == 0) {	
		hipMemcpy(avg_array, dev_array2, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}
	else
	{
		hipMemcpy(avg_array, dev_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}

	double avg = avg_array[0];
	free( avg_array );
	hipFree( dev_avg_array );
	
	double *std_array = (double *) malloc(nblocks * sizeof(double));
	double *dev_std_array;
	hipMalloc((void**)&dev_std_array, nblocks * sizeof(double));

	hipMemcpy(dev_array, array, arr_size, hipMemcpyHostToDevice);

	std_start = clock();

	size = num_elem;
	nblocks = (long int)ceil((double)num_elem / (double)tpb);
	count = 0;
	while (nblocks >= 1) {
		//printf("Calling kernel_std\n");
		//printf("nblocks: %ld\n", nblocks);
		if (count % 2 == 0) {
			kernel_std<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array, dev_array2, num_elem, avg, count, nblocks);
		}
		else
		{
			kernel_std<<<nblocks, tpb, tpb * sizeof(double)>>>(dev_array2, dev_array, num_elem, avg, count, nblocks);
		}
		hipDeviceSynchronize();
		if (nblocks == 1) { break; }
		count++;
		size = nblocks;
		nblocks = (long int)ceil((double)size / (double)tpb);
	}
	std_end = clock();
	printf("Time for Std: %f\n", (((double)avg_end - (double)avg_start) + ((double)std_end - (double)std_start)) / CLOCKS_PER_SEC);


	if (count % 2 == 0) {	
		hipMemcpy(std_array, dev_array2, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}
	else
	{
		hipMemcpy(std_array, dev_array, nblocks * sizeof(double), hipMemcpyDeviceToHost);
	}

	printf( "Std: %f\n", std_array[0] );

	free( std_array );
	hipFree( dev_array );
	hipFree( dev_array2 );
	hipFree( dev_std_array );
}

